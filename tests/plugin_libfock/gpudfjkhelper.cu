#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hip/hip_runtime.h>
#include"gpudfjkhelper.h"
#include"blas.h"
#ifdef _OPENMP
   #include<omp.h>
#endif

using namespace psi;

namespace psi{

/**
 * constructor
 */
GPUDFJKHelper::GPUDFJKHelper(){}

/**
 * destructor
 */
GPUDFJKHelper::~GPUDFJKHelper(){}

/**
 * check for errors following cuda calls
 */
inline void GPUDFJKHelper::Check_CUDA_Error(FILE*fp,const char *message){
  hipError_t error = hipGetLastError();
  if (error!=hipSuccess) {
     fprintf(fp,"\n  ERROR: %s: %s\n\n", message, hipGetErrorString(error) );
     fflush(fp);
     exit(-1);
  }
}

/**
 * free host memory
 */
void GPUDFJKHelper::Finalize(){
  // free cpu memory and reset each device
  #pragma omp parallel for schedule (static)
  for (int i=0; i<num_gpus; i++){
      int nthreads = omp_get_num_threads();
      int thread = 1;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif
      if (i<nthreads){
         hipHostFree(tmp[thread]);
         Check_CUDA_Error(stdout,"hipHostFree");
         hipDeviceReset();
         Check_CUDA_Error(stdout,"hipDeviceReset");
      }
  }
}

/**
 * initialize temporary array for mapping cpu memory before transfer
 * to gpu.  also, set number of gpus. also, initialize cublas
 */
void GPUDFJKHelper::Initialize(int max_rows,int max_nocc,int nbf){

  // allocate cpu memory
  int gpu_id; 
  hipGetDevice(&gpu_id);
  struct hipDeviceProp_t cudaProp;
  hipGetDeviceProperties( &cudaProp,gpu_id );

  // memory available to the device (less a little extra)
  double memory = cudaProp.totalGlobalMem/8. - 200.*1024*1024/8.;
  int dim = (int)memory;

  cublasInit();
  Check_CUDA_Error(stdout,"cublasInit");
  hipGetDeviceCount(&num_gpus);
  Check_CUDA_Error(stdout,"hipGetDeviceCount");
  hipDeviceReset();
  Check_CUDA_Error(stdout,"hipDeviceReset");
  tmp = (double**)malloc(num_gpus*sizeof(double*));

  // initialize each device and device memory:
  #pragma omp parallel for schedule (static)
  for (int i=0; i<num_gpus; i++){
      int nthreads = omp_get_num_threads();
      int thread = 1;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif
      if (i<nthreads){
         hipSetDevice(thread);
         Check_CUDA_Error(stdout,"hipSetDevice");
         hipHostMalloc((void**)&tmp[thread],dim*sizeof(double));
         Check_CUDA_Error(stdout,"hipHostMalloc");
     }
  }

}


/**
 * dgemm assuming no tiling is necessary
 */
void GPUDFJKHelper::GPU_DGEMM(char transa,char transb,int m,int n,int k,double alpha,double*A,int lda,double*B,int ldb,double beta,double*C,int ldc){
  double*gpuA,*gpuB,*gpuC;
  hipMalloc((void**)&gpuA,m*k*sizeof(double));
  hipMalloc((void**)&gpuB,n*k*sizeof(double));
  hipMalloc((void**)&gpuC,m*n*sizeof(double));
  hipMemcpy(gpuA,A,m*k*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(gpuB,B,n*k*sizeof(double),hipMemcpyHostToDevice);
  hipblasDgemm(transa,transb,m,n,k,alpha,gpuA,lda,gpuB,ldb,beta,gpuC,ldc);
  hipMemcpy(C,gpuC,m*n*sizeof(double),hipMemcpyDeviceToHost);
  hipFree(gpuA);
  hipFree(gpuB);
  hipFree(gpuC);
}
/**
 * dgemm using a 2-dimensional tile.  this guy is also threaded.
 */
void GPUDFJKHelper::GPU_DGEMM_2DTile(char transa,char transb,int m,int n,int k,double alpha,double*A,int lda,double*B,int ldb,double beta,double*C,int ldc,int thread){

  // cpu threads:
  if (thread>=num_gpus){
     F_DGEMM(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
     return;
  } 

  int nthreads = omp_get_num_threads();

  struct hipDeviceProp_t cudaProp;
  int gpu_id;
  hipGetDevice(&gpu_id);
  hipGetDeviceProperties( &cudaProp,gpu_id );

  // memory available to the device (less a little extra)
  double memory = cudaProp.totalGlobalMem/8. - 200.*1024*1024/8.;

  // test not having enough memory:
  //memory = (m*n+k*(m+n))/7.;
     
  // determine what tiling should be:
  int ntilesN,ntilesM,ntilesK,tilesizeK,tilesizeN,tilesizeM,lasttileK,lasttileN,lasttileM;
  tilesizeN = n;
  tilesizeM = m;
  tilesizeK = k;
  ntilesM=ntilesN=ntilesK=1;
  while(tilesizeN*tilesizeM+tilesizeK*(tilesizeN+tilesizeM)>(int)memory){
     if (tilesizeN>tilesizeM){
        if (tilesizeN>tilesizeK){
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
     }
     else{
        if (tilesizeM>tilesizeK){
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
        else{
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
     }
  }
  lasttileN = n - (ntilesN-1)*tilesizeN;
  lasttileM = m - (ntilesM-1)*tilesizeM;
  lasttileK = k - (ntilesK-1)*tilesizeK;

  // allocate gpu memory
  double*gpuA,*gpuB,*gpuC;
  hipMalloc((void**)&gpuA,tilesizeM*tilesizeK*sizeof(double));
  hipMalloc((void**)&gpuB,tilesizeN*tilesizeK*sizeof(double));
  hipMalloc((void**)&gpuC,tilesizeM*tilesizeN*sizeof(double));

  int tm,tn,tk,i,j;

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (i=0; i<n*ldc; i++) C[i] *= beta;

  for (tm=0; tm<ntilesM-1; tm++){
      for (tn=0; tn<ntilesN-1; tn++){
          for (tk=0; tk<ntilesK-1; tk++){

              for (i=0; i<tilesizeM; i++){
                  F_DCOPY(tilesizeK,A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizeK,1);
              }
              hipMemcpy(gpuA,tmp[thread],tilesizeM*tilesizeK*sizeof(double),hipMemcpyHostToDevice);
              for (i=0; i<tilesizeN; i++){
                  F_DCOPY(tilesizeK,B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizeK,1);
              }
              hipMemcpy(gpuB,tmp[thread],tilesizeN*tilesizeK*sizeof(double),hipMemcpyHostToDevice);
              hipblasDgemm(transa,transb,tilesizeM,tilesizeN,tilesizeK,alpha,gpuA,tilesizeK,gpuB,tilesizeK,0.0,gpuC,tilesizeM);
              hipMemcpy(tmp[thread],gpuC,tilesizeN*tilesizeM*sizeof(double),hipMemcpyDeviceToHost);
              for (j=0; j<tilesizeN; j++){
                  F_DAXPY(tilesizeM,1.0,tmp[thread]+j*tilesizeM,1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
              }
          } // end of tiles over k

          // last tile of k
          tk = ntilesK-1;

          for (i=0; i<tilesizeM; i++){
              F_DCOPY(lasttileK,A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*lasttileK,1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizeM*lasttileK*sizeof(double),hipMemcpyHostToDevice);
          for (i=0; i<tilesizeN; i++){
              F_DCOPY(lasttileK,B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*lasttileK,1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizeN*lasttileK*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizeM,tilesizeN,lasttileK,alpha,gpuA,lasttileK,gpuB,lasttileK,0.0,gpuC,tilesizeM);
          hipMemcpy(tmp[thread],gpuC,tilesizeN*tilesizeM*sizeof(double),hipMemcpyDeviceToHost);
          for (j=0; j<tilesizeN; j++){
              F_DAXPY(tilesizeM,1.0,tmp[thread]+j*tilesizeM,1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
          }

      } // end of tiles over n

  } // end of tiles over m

  // last tiles of m and n:
  tm = ntilesM-1;
  for (tn=0; tn<ntilesN-1; tn++){
      for (tk=0; tk<ntilesK-1; tk++){

          for (i=0; i<lasttileM; i++){
              F_DCOPY(tilesizeK,A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizeK,1);
          }
          hipMemcpy(gpuA,tmp[thread],lasttileM*tilesizeK*sizeof(double),hipMemcpyHostToDevice);
          for (i=0; i<tilesizeN; i++){
              F_DCOPY(tilesizeK,B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizeK,1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizeN*tilesizeK*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,lasttileM,tilesizeN,tilesizeK,alpha,gpuA,tilesizeK,gpuB,tilesizeK,0.0,gpuC,lasttileM);
          hipMemcpy(tmp[thread],gpuC,tilesizeN*lasttileM*sizeof(double),hipMemcpyDeviceToHost);
          for (j=0; j<tilesizeN; j++){
              F_DAXPY(lasttileM,1.0,tmp[thread]+j*lasttileM,1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
          }
      } // end of tiles over k

      // last tile of k
      tk = ntilesK-1;
 
      for (i=0; i<lasttileM; i++){
          F_DCOPY(lasttileK,A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*lasttileK,1);
      }
      hipMemcpy(gpuA,tmp[thread],lasttileM*lasttileK*sizeof(double),hipMemcpyHostToDevice);
      for (i=0; i<tilesizeN; i++){
          F_DCOPY(lasttileK,B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*lasttileK,1);
      }
      hipMemcpy(gpuB,tmp[thread],tilesizeN*lasttileK*sizeof(double),hipMemcpyHostToDevice);
      hipblasDgemm(transa,transb,lasttileM,tilesizeN,lasttileK,alpha,gpuA,lasttileK,gpuB,lasttileK,0.0,gpuC,lasttileM);
      hipMemcpy(tmp[thread],gpuC,tilesizeN*lasttileM*sizeof(double),hipMemcpyDeviceToHost);
      for (j=0; j<tilesizeN; j++){
          F_DAXPY(lasttileM,1.0,tmp[thread]+j*lasttileM,1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  } // end of tiles over n


  tn = ntilesN-1; 
  for (tm=0; tm<ntilesM-1; tm++){
      for (tk=0; tk<ntilesK-1; tk++){

          for (i=0; i<tilesizeM; i++){
              F_DCOPY(tilesizeK,A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizeK,1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizeM*tilesizeK*sizeof(double),hipMemcpyHostToDevice);
          for (i=0; i<lasttileN; i++){
              F_DCOPY(tilesizeK,B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizeK,1);
          }
          hipMemcpy(gpuB,tmp[thread],lasttileN*tilesizeK*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizeM,lasttileN,tilesizeK,alpha,gpuA,tilesizeK,gpuB,tilesizeK,0.0,gpuC,tilesizeM);
          hipMemcpy(tmp[thread],gpuC,lasttileN*tilesizeM*sizeof(double),hipMemcpyDeviceToHost);
          for (j=0; j<lasttileN; j++){
              F_DAXPY(tilesizeM,1.0,tmp[thread]+j*tilesizeM,1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
          }
      } // end of tiles over k

      // last tile of k
      tk = ntilesK-1;

      for (i=0; i<tilesizeM; i++){
          F_DCOPY(lasttileK,A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*lasttileK,1);
      }
      hipMemcpy(gpuA,tmp[thread],tilesizeM*lasttileK*sizeof(double),hipMemcpyHostToDevice);
      for (i=0; i<lasttileN; i++){
          F_DCOPY(lasttileK,B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*lasttileK,1);
      }
      hipMemcpy(gpuB,tmp[thread],lasttileN*lasttileK*sizeof(double),hipMemcpyHostToDevice);
      hipblasDgemm(transa,transb,tilesizeM,lasttileN,lasttileK,alpha,gpuA,lasttileK,gpuB,lasttileK,0.0,gpuC,tilesizeM);
      hipMemcpy(tmp[thread],gpuC,lasttileN*tilesizeM*sizeof(double),hipMemcpyDeviceToHost);
      for (j=0; j<lasttileN; j++){
          F_DAXPY(tilesizeM,1.0,tmp[thread]+j*tilesizeM,1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  } // end of tiles over m

  tn = ntilesN-1; 
  tm = ntilesM-1; 
  for (tk=0; tk<ntilesK-1; tk++){

      for (i=0; i<lasttileM; i++){
          F_DCOPY(tilesizeK,A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizeK,1);
      }
      hipMemcpy(gpuA,tmp[thread],lasttileM*tilesizeK*sizeof(double),hipMemcpyHostToDevice);
      for (i=0; i<lasttileN; i++){
          F_DCOPY(tilesizeK,B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizeK,1);
      }
      hipMemcpy(gpuB,tmp[thread],lasttileN*tilesizeK*sizeof(double),hipMemcpyHostToDevice);
      hipblasDgemm(transa,transb,lasttileM,lasttileN,tilesizeK,alpha,gpuA,tilesizeK,gpuB,tilesizeK,0.0,gpuC,lasttileM);
      hipMemcpy(tmp[thread],gpuC,lasttileN*lasttileM*sizeof(double),hipMemcpyDeviceToHost);
      for (j=0; j<lasttileN; j++){
          F_DAXPY(lasttileM,1.0,tmp[thread]+j*lasttileM,1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  } // end of tiles over k

  // last tile of k
  tk = ntilesK-1;

  for (i=0; i<lasttileM; i++){
      F_DCOPY(lasttileK,A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*lasttileK,1);
  }
  hipMemcpy(gpuA,tmp[thread],lasttileM*lasttileK*sizeof(double),hipMemcpyHostToDevice);
  for (i=0; i<lasttileN; i++){
      F_DCOPY(lasttileK,B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*lasttileK,1);
  }
  hipMemcpy(gpuB,tmp[thread],lasttileN*lasttileK*sizeof(double),hipMemcpyHostToDevice);
  hipblasDgemm(transa,transb,lasttileM,lasttileN,lasttileK,alpha,gpuA,lasttileK,gpuB,lasttileK,0.0,gpuC,lasttileM);
  hipMemcpy(tmp[thread],gpuC,lasttileN*lasttileM*sizeof(double),hipMemcpyDeviceToHost);
  for (j=0; j<lasttileN; j++){
      F_DAXPY(lasttileM,1.0,tmp[thread]+j*lasttileM,1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
  }
  hipFree(gpuA);
  hipFree(gpuB);
  hipFree(gpuC);

}
}//end of namespace
