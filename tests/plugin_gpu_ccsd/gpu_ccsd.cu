#include "hip/hip_runtime.h"
#include"psi4-dec.h"
#include<libmints/wavefunction.h>
#include<libmints/vector.h>
#include<libpsio/psio.hpp>
#include<sys/times.h>

#include"blas.h"
#include"gpu_ccsd.h"
#include"gpu_kernels.h"

inline void Check_CUDA_Error(FILE*fp,const char *message){
  hipError_t error = hipGetLastError();
  if (error!=hipSuccess) {
     fprintf(fp,"\n  ERROR: %s: %s\n\n", message, hipGetErrorString(error) );
     fflush(fp);
     exit(-1);
  }
}

using namespace psi;
using namespace boost;

namespace psi{
  void ReadTEIs(double*tei,Options&options);
}

// position in a symmetric packed matrix
long Position(int i,int j){
  if (i<j){
    return ((j*(j+1))>>1)+i;
  }
  return ((i*(i+1))>>1)+j;
}

namespace psi{
GPUCoupledCluster::GPUCoupledCluster()
{}
GPUCoupledCluster::~GPUCoupledCluster()
{}

void GPUCoupledCluster::WriteBanner(Options &options){
  fflush(outfile);
  fprintf(outfile,"\n\n");
  fprintf(outfile, "        *******************************************************\n");
  fprintf(outfile, "        *                                                     *\n");
  fprintf(outfile, "        *                        CCSD                         *\n");
  fprintf(outfile, "        *           Coupled-Cluster Singles Doubles           *\n");
  fprintf(outfile, "        *                                                     *\n");
  fprintf(outfile, "        *                   Eugene DePrince                   *\n");
  fprintf(outfile, "        *                                                     *\n");
  fprintf(outfile, "        *******************************************************\n");
  fprintf(outfile,"\n\n");
  fflush(outfile);
}

/*================================================================
  
  Initialize:
  set essential variables (ndocc...).  Read 1- and 2-electron
  integrals into core.  Sort and write integrals.
  
================================================================*/
void GPUCoupledCluster::Initialize(Options &options){

  // grab the reference wave function and its parameters
  boost::shared_ptr<Wavefunction> ref = Process::environment.reference_wavefunction();

  if (ref.get() !=NULL){
     escf    = Process::environment.globals["SCF TOTAL ENERGY"];
     nirreps = ref->nirrep();
     sorbs   = ref->nsopi();
     orbs    = ref->nmopi();
     docc    = ref->doccpi();
     fzc     = ref->frzcpi();
     fzv     = ref->frzvpi();
  }
  if (nirreps>1){
     //throw PsiException("plugin_gpu_ccsd requires symmetry c1",__FILE__,__LINE__);
  }
  nso = nmo = ndocc = nvirt = nfzc = nfzv = 0;
  int full=0;
  for (int h=0; h<nirreps; h++){
      nfzc   += fzc[h];
      nfzv   += fzv[h];
      nso    += sorbs[h];
      full   += orbs[h];
      nmo    += orbs[h]-fzc[h]-fzv[h];
      ndocc  += docc[h];//-fzc[h];
  }
  ndoccact = ndocc - nfzc;
  nvirt  = nmo - ndoccact;

  if (nvirt<ndoccact){
     throw PsiException("plugin_gpu_ccsd requires more virtual orbitals than active doubly occupied orbitals",__FILE__,__LINE__);
  }

  // get paramters from input 
  conv    = options.get_double("R_CONVERGENCE");
  maxiter = options.get_int("MAXITER");
  maxdiis = options.get_int("DIIS_MAX_VECS");

  // memory is from process::environment, but can override that
  memory = Process::environment.get_memory();
  if (options["MEMORY"].has_changed()){
     memory  = options.get_int("MEMORY");
     memory *= (long int)1024*1024;
  }

  // SCS MP2 and CCSD
  emp2_os_fac = options.get_double("MP2_SCALE_OS");
  emp2_ss_fac = options.get_double("MP2_SCALE_SS");
  eccsd_os_fac = options.get_double("CC_SCALE_OS");
  eccsd_ss_fac = options.get_double("CC_SCALE_SS");

  //boost::shared_ptr<Matrix> Ca = ref->Ca();

  nmotemp = full;//Ca->colspi()[0];

  // orbital energies
  /*eps_test = ref->epsilon_a();
  int i;
  double*tmpeps = eps_test->pointer();
  eps = (double*)malloc(nmo*sizeof(double));
  for (i=0; i<nmo; i++) eps[i] = tmpeps[i+nfzc];
  eps_test.reset();*/

  // orbital energies
  eps = (double*)malloc(nmo*sizeof(double));
  int count=0;
  for (int h=0; h<nirreps; h++){
      eps_test = ref->epsilon_a();
      for (int norb = fzc[h]; norb<docc[h]; norb++){
          eps[count++] = eps_test->get(h,norb);
      }
  }
  for (int h=0; h<nirreps; h++){
      eps_test = ref->epsilon_a();
      for (int norb = docc[h]; norb<orbs[h]-fzv[h]; norb++){
          eps[count++] = eps_test->get(h,norb);
      }
  }
  eps_test.reset();


  // so->mo tei transformation (no, we're just reading from disk)
  struct tms total_tmstime;
  const long clk_tck = sysconf(_SC_CLK_TCK);

  double time_start,user_start,sys_start,time_stop,user_stop,sys_stop;

  // sort integrals and write them to disk
  times(&total_tmstime);
  time_start = time(NULL);
  user_start = ((double) total_tmstime.tms_utime)/clk_tck;
  sys_start  = ((double) total_tmstime.tms_stime)/clk_tck;

  int ntri = nmotemp*(nmotemp+1)/2;
  ntri = ntri*(ntri+1)/2;
  tei = (double*)malloc(sizeof(double)*ntri);
  memset((void*)tei,'\0',ntri*sizeof(double));

  // read integrals 
  ReadTEIs(tei,options);

  // sort integrals and write them to disk
  WriteIntegrals(tei);

  times(&total_tmstime);
  time_stop = time(NULL);
  user_stop = ((double) total_tmstime.tms_utime)/clk_tck;
  sys_stop  = ((double) total_tmstime.tms_stime)/clk_tck;

  fprintf(outfile,"  Time for integral sort:           %6.2lf s (user)\n",user_stop-user_start);
  fprintf(outfile,"                                    %6.2lf s (system)\n",sys_stop-sys_start);
  fprintf(outfile,"                                    %6d s (total)\n",(int)time_stop-(int)time_start);

  // free teis
  free(tei);

  // t2 is in core
  t2_on_disk = false;
}

/*===================================================================

  solve ccsd equations

===================================================================*/
PsiReturnType GPUCoupledCluster::CCSDIterations(Options&options){

  struct tms total_tmstime;
  const long clk_tck = sysconf(_SC_CLK_TCK);
  time_t iter_start,iter_stop,time_start,time_stop;
  double user_start,user_stop,sys_start,sys_stop;

  int count,j,replace_diis_iter,diis_iter,iter;
  int o = ndoccact;
  int v = nvirt;
  int oo1o2 = o*(o+1)/2;
  int vv1o2 = v*(v+1)/2;

  diis_iter=iter=0;
  replace_diis_iter=1;
  double nrm=1.0;
  double Eold=1.0e9;
  eccsd=0.0;

  // get device parameters, set nblocks,nthreads,dimgrid
  CudaInit();
  // TODO: once nvcc can be used with psi, add dimgrid to GPUCoupledCluster class
  dim3 dimgrid (nblocks,num);

  // define tiling for v^4 and ov^3 diagrams on the gpu
  DefineTiling();

  // allocate memory on the gpu
  AllocateGPUMemory();

  hipStream_t stream;
  hipEvent_t estart,estop;
  hipEventCreate(&estart);
  hipEventCreate(&estop);

  fprintf(outfile,"\n");
  fprintf(outfile,
    "  Begin singles and doubles coupled cluster iterations\n\n");
  fprintf(outfile,
    "   Iter  DIIS          Energy       d(Energy)          |d(T)|     time\n");
  fflush(outfile);

  boost::shared_ptr<PSIO> psio(new PSIO());

  // start timing the iterations
  times(&total_tmstime);
  time_start = time(NULL);
  user_start = ((double) total_tmstime.tms_utime)/clk_tck;
  sys_start  = ((double) total_tmstime.tms_stime)/clk_tck;

  //TCEPA();
  while(iter<maxiter && nrm>conv){
      iter_start = time(NULL);

      memset((void*)w1,'\0',o*v*sizeof(double));
      memset((void*)wb,'\0',o*o*v*v*sizeof(double));
      if (iter>0){


         stream = NULL;

         // copy amplitudes to the device
         hipEventRecord(estart,stream);
             hipMemcpyAsync(gput1,t1,sizeof(double)*o*v,hipMemcpyHostToDevice,stream);
             hipMemcpyAsync(gput2,tb,sizeof(double)*o*o*v*v,hipMemcpyHostToDevice,stream);
         hipEventRecord(estop,stream);

         while( hipEventQuery(estop) == hipErrorNotReady );

         hipEventRecord(estart,stream);

            //==========================================================
            //
            // I(ij,kl)
            //
            //==========================================================
            hipMemcpyAsync(gpuw,E2klcd_1,sizeof(double)*o*o*v*v,hipMemcpyHostToDevice,stream);
            GPUt2Plust1_and_E2klcd3<<<dimgrid,nthreads>>>(o,v,gput2,gput1,gpuw,gpuv);

            // Build and use I2(ij,kl)
            hipMemcpyAsync(gpuw,E2ijkl,sizeof(double)*o*o*o*o,hipMemcpyHostToDevice,stream);
            hipblasDgemm('n','n',o*o,o*o,v*v,1.0,gput2,o*o,gpuv,v*v,1.0,gpuw,o*o);

            hipMemcpyAsync(gpuv,E2ijakCopy,sizeof(double)*o*o*o*v,hipMemcpyHostToDevice,stream);
            hipblasDgemm('n','n',o,o*o*o,v,2.0,gput1,o,gpuv,v,1.0,gpuw,o);

            hipblasDgemm('n','n',o*o,v*v,o*o,0.5,gpuw,o*o,gput2,o*o,0.0,gputempw,o*o);

            AddPermutedOnGPU<<<dimgrid,nthreads>>>(o,v,gputempw,gpuw);

            //==========================================================
            //
            // I(ia,jk)
            //
            //==========================================================

            // Build and use I2(ia,jk)
            hipMemcpyAsync(gputempw,E2ijak2,sizeof(double)*o*o*o*v,hipMemcpyHostToDevice,stream);

            for (j=0; j<novtiles-1; j++){
                hipMemcpyAsync(gpuv,E2abci+j*v*v*ovtilesize,sizeof(double)*v*v*ovtilesize,hipMemcpyHostToDevice,stream);
                hipblasDgemm('n','n',o*o,ovtilesize,v*v,1.0,gput2,o*o,gpuv,v*v,1.0,gputempw+j*o*o*ovtilesize,o*o);
            }
            hipMemcpyAsync(gpuv,E2abci+j*v*v*ovtilesize,sizeof(double)*v*v*lastovtile,hipMemcpyHostToDevice,stream);
            hipblasDgemm('n','n',o*o,lastovtile,v*v,1.0,gput2,o*o,gpuv,v*v,1.0,gputempw+j*o*o*ovtilesize,o*o);

            hipblasDgemm('n','n',o*o*v,v,o,-1.0,gputempw,o*o*v,gput1,o,0.0,gpuv,o*o*v);

            GPUFill_I2iajk_and_c2Sym1<<<dimgrid,nthreads>>>(o,v,gpuv,gpuw,gput2,gputempw);

            //==========================================================
            //
            // t(ij,ef) v(ef,ab) - scuseria/jansen version
            //
            //==========================================================
            for (j=0; j<ntiles-1; j++){
                hipMemcpyAsync(gpuv,Symabcd1+j*tilesize*vv1o2,tilesize*vv1o2*sizeof(double),hipMemcpyHostToDevice,stream);
                hipblasDgemm('n','n',oo1o2,tilesize,vv1o2,0.5,gputempw,oo1o2,gpuv,vv1o2,0.0,gput2+j*tilesize*oo1o2,oo1o2);
            }
            j=ntiles-1;
            hipMemcpyAsync(gpuv,Symabcd1+j*tilesize*vv1o2,lasttile*vv1o2*sizeof(double),hipMemcpyHostToDevice,stream);
            hipblasDgemm('n','n',oo1o2,lasttile,vv1o2,0.5,gputempw,o*(o+1)/2,gpuv,vv1o2,0.0,gput2+j*tilesize*oo1o2,oo1o2);
            GPUSymmAdd1<<<dimgrid,nthreads>>>(o,v,gput2,gpuw);

            hipMemcpyAsync(gput2,tb,sizeof(double)*o*o*v*v,hipMemcpyHostToDevice,stream);
            GPUc2Sym2_onefunction<<<dimgrid,nthreads>>>(o,v,gput2,gput1,gputempw);
            for (j=0; j<ntiles-1; j++){
                hipMemcpyAsync(gpuv,Symabcd2+j*tilesize*vv1o2,tilesize*vv1o2*sizeof(double),hipMemcpyHostToDevice,stream);
                hipblasDgemm('n','n',oo1o2,tilesize,vv1o2,0.5,gputempw,oo1o2,gpuv,vv1o2,0.0,gput2+j*tilesize*oo1o2,oo1o2);
            }
            j=ntiles-1;
            hipMemcpyAsync(gpuv,Symabcd2+j*tilesize*vv1o2,lasttile*v*(v+1)/2*sizeof(double),hipMemcpyHostToDevice,stream);
            hipblasDgemm('n','n',oo1o2,lasttile,vv1o2,0.5,gputempw,oo1o2,gpuv,vv1o2,0.0,gput2+j*tilesize*oo1o2,oo1o2);

            GPUSymmAdd2<<<dimgrid,nthreads>>>(o,v,gput2,gpuw);
            hipMemcpyAsync(tempu,gpuw,sizeof(double)*o*o*v*v,hipMemcpyDeviceToHost,stream);

         // end of gpu section
         hipEventRecord(estop,stream);

         // evaluate a few diagrams on the cpu while gpu runs
         CPU_t1_vmeni();
         CPU_t1_vmaef();
         CPU_I2p_abci_refactored();

         count=0;
         while( hipEventQuery(estop) == hipErrorNotReady )count++;
         //if (count==0)fprintf(outfile,"          Warning: CPU time exceeds GPU vabcd stream time\n");
            
         F_DAXPY(o*o*v*v,1.0,tempu,1,wb,1);

         // reset cublas stream to null stream
         cublasSetKernelStream(NULL);

         // build I2(ia,bj), which contributes to t2
         cublasSetKernelStream(stream);
         hipEventRecord(estart,stream);
             hipMemcpyAsync(gpuw,E2klcd_1,sizeof(double)*o*o*v*v,hipMemcpyHostToDevice,stream);

             hipMemcpyAsync(gput2,tb,sizeof(double)*o*o*v*v,hipMemcpyHostToDevice,stream);
             GPUt2Plus2t1<<<dimgrid,nthreads>>>(o,v,gpuv,gput2,gput1,gputempw);

             hipMemcpyAsync(gput2,E2klcd_1,sizeof(double)*o*o*v*v,hipMemcpyHostToDevice,stream);
             hipblasDgemm('n','n',o*v,o*v,o*v,-0.5,gpuv,o*v,gpuw,o*v,1.0,gput2,o*v);

             GPUv2MinusHalfv2<<<dimgrid,nthreads>>>(o,v,gpuv,gpuw);
             hipblasDgemm('n','n',o*v,o*v,o*v,1.0,gputempw,o*v,gpuv,o*v,1.0,gput2,o*v);

             for (j=0; j<nov2tiles-1; j++){
                 hipMemcpyAsync(gpuv,E2abci+j*v*ov2tilesize,sizeof(double)*v*ov2tilesize,hipMemcpyHostToDevice,stream);
                 hipblasDgemm('n','n',o,ov2tilesize,v,1.0,gput1,o,gpuv,v,0.0,gpuw+j*o*ov2tilesize,o);
             }
             j=nov2tiles-1;
             hipMemcpyAsync(gpuv,E2abci+j*v*ov2tilesize,sizeof(double)*v*lastov2tile,hipMemcpyHostToDevice,stream);
             hipblasDgemm('n','n',o,lastov2tile,v,1.0,gput1,o,gpuv,v,0.0,gpuw+j*o*ov2tilesize,o);

             GPUPermute_iabj_to_aijb<<<dimgrid,nthreads>>>(o,v,gpuw,gpuv);

             hipMemcpyAsync(gpuw,E2ijakCopy,sizeof(double)*o*o*o*v,hipMemcpyHostToDevice,stream);
             hipblasDgemm('n','n',o*o*v,v,o,-1.0,gpuw,o*o*v,gput1,o,1.0,gpuv,o*o*v);

             GPUFill_I2iabj<<<dimgrid,nthreads>>>(o,v,gput2,gpuv,gpuw);

             // use I2(ia,bj)
             GPU2t2Minust2<<<dimgrid,nthreads>>>(o,v,gpuv,gputempw);
             hipblasDgemm('n','t',o*v,o*v,o*v,1.0,gpuw,o*v,gpuv,o*v,0.0,gput2,o*v);

             GPUFill_t2_I2iajb<<<dimgrid,nthreads>>>(o,v,gput2,gpuw);
             hipMemcpyAsync(tempu,gpuw,sizeof(double)*o*o*v*v,hipMemcpyDeviceToHost,stream);
         hipEventRecord(estop,stream);

         // read integrals and do one small diagram on the cpu
         CPU_I1ab();

         count=0;
         while( hipEventQuery(estop) == hipErrorNotReady )count++;
         //if (count==0) fprintf(outfile,"          Warning: CPU time exceeds GPU I(ia,bj) stream time\n");

         psio->open(PSIF_ABCI4,PSIO_OPEN_OLD);
         psio->read_entry(PSIF_ABCI4,"E2abci4",(char*)&E2abci[0],v*v*v*o*sizeof(double));
         psio->close(PSIF_ABCI4,1);

         cublasSetKernelStream(NULL);
         F_DAXPY(o*o*v*v,1.0,tempu,1,wb,1);

         // build I2(ia,jb), which contributes to t2
         cublasSetKernelStream(stream);
         hipEventRecord(estart,stream);
             hipMemcpyAsync(gpuw,E2klcd_1,sizeof(double)*o*o*v*v,hipMemcpyHostToDevice,stream);
             GPUt2Plus2t1_and_E2klcd2<<<dimgrid,nthreads>>>(o,v,gpuv,gputempw,gput1,gpuw,gput2);

             hipMemcpyAsync(gpuw,E2akjc_2,sizeof(double)*o*o*v*v,hipMemcpyHostToDevice,stream);
             hipblasDgemm('n','n',o*v,o*v,o*v,-0.5,gpuv,o*v,gput2,o*v,1.0,gpuw,o*v);

             for (j=0; j<nov2tiles-1; j++){
                 hipMemcpyAsync(gpuv,E2abci+j*v*ov2tilesize,sizeof(double)*v*ov2tilesize,hipMemcpyHostToDevice,stream);
                 hipblasDgemm('n','n',o,ov2tilesize,v,1.0,gput1,o,gpuv,v,0.0,gput2+j*o*ov2tilesize,o);

             }
             j=nov2tiles-1;
             hipMemcpyAsync(gpuv,E2abci+j*v*ov2tilesize,sizeof(double)*v*lastov2tile,hipMemcpyHostToDevice,stream);
             hipblasDgemm('n','n',o,lastov2tile,v,1.0,gput1,o,gpuv,v,0.0,gput2+j*o*ov2tilesize,o);
             GPUFillI2iajb1<<<dimgrid,nthreads>>>(o,v,gput2,gpuw);

             hipMemcpyAsync(gpuv,E2ijak3,sizeof(double)*o*o*o*v,hipMemcpyHostToDevice,stream);
             hipblasDgemm('n','n',o*o*v,v,o,-1.0,gpuv,o*o*v,gput1,o,0.0,gput2,o*o*v);
             GPUFillI2iajb2<<<dimgrid,nthreads>>>(o,v,gput2,gpuw);

             // Use I2(ia,jb)
             hipblasDgemm('n','t',o*v,o*v,o*v,-1.0,gpuw,o*v,gputempw,o*v,0.0,gput2,o*v);
             GPUFill_t2_I2iabj1<<<dimgrid,nthreads>>>(o,v,gput2,gpuv);

             // Use I2(ia,jb) again
             GPUPermute_tikbc<<<dimgrid,nthreads>>>(gputempw,gput2,o,v);

             hipblasDgemm('n','t',o*v,o*v,o*v,-1.0,gpuw,o*v,gput2,o*v,0.0,gputempw,o*v);
             GPUFill_t2_I2iabj2<<<dimgrid,nthreads>>>(o,v,gputempw,gpuv);

             hipMemcpyAsync(tempu,gpuv,sizeof(double)*o*o*v*v,hipMemcpyDeviceToHost,stream);
         hipEventRecord(estop,stream);

         // do some small diagrams on the cpu
         CPU_t1_vmeai();
         CPU_I1pij_I1ia_lessmem();

         count=0;
         while( hipEventQuery(estop) == hipErrorNotReady )count++;
         //if (count==0) fprintf(outfile,"          Warning: CPU I1pij+I1ia time exceeds GPU I(ia,jb) stream time\n");

         cublasSetKernelStream(NULL);

         F_DAXPY(o*o*v*v,1.0,tempu,1,wb,1);

         // TODO:  find a better place to do this.  refill E2abci
         psio->open(PSIF_ABCI,PSIO_OPEN_OLD);
         psio->read_entry(PSIF_ABCI,"E2abci",(char*)&E2abci[0],v*v*v*o*sizeof(double));
         psio->close(PSIF_ABCI,1);
      }

      // update the amplitudes
      Eold = eccsd;
      UpdateT1(iter);
      UpdateT2(iter);

      // add vector to list for diis
      DIISOldVector(iter,diis_iter,replace_diis_iter);

      // diis error vector and convergence check
      nrm = DIISErrorVector(diis_iter,replace_diis_iter,iter);

      // diis extrapolation
      if (diis_iter>1){
         if (diis_iter<maxdiis) DIIS(diisvec,diis_iter,o*o*v*v+o*v);
         else                   DIIS(diisvec,maxdiis,o*o*v*v+o*v);
         DIISNewAmplitudes(diis_iter);
      }
      eccsd = CheckEnergy();

      if (diis_iter<=maxdiis) diis_iter++;
      else if (replace_diis_iter<maxdiis) replace_diis_iter++;
      else replace_diis_iter = 1;

      iter_stop = time(NULL);
      fprintf(outfile,"  %5i   %i %i %15.10f %15.10f %15.10f %8d\n",
            iter,diis_iter-1,replace_diis_iter,eccsd,eccsd-Eold,nrm,(int)iter_stop-(int)iter_start);
      fflush(outfile);
      iter++;
      if (iter==1){
         emp2 = eccsd;
         SCS_MP2();
      }
  }
  times(&total_tmstime);
  time_stop = time(NULL);
  user_stop = ((double) total_tmstime.tms_utime)/clk_tck;
  sys_stop  = ((double) total_tmstime.tms_stime)/clk_tck;
  psio.reset();

  if (iter==maxiter){
     throw PsiException("  CCSD iterations did not converge.",__FILE__,__LINE__);
  }

  SCS_CCSD();

  fprintf(outfile,"\n");
  fprintf(outfile,"  CCSD iterations converged!\n");
  fprintf(outfile,"\n");
  if (options.get_bool("SCS_MP2")){
     fprintf(outfile,"        OS SCS-MP2 correlation energy:  %20.12lf\n",emp2_os*emp2_os_fac);
     fprintf(outfile,"        SS SCS-MP2 correlation energy:  %20.12lf\n",emp2_ss*emp2_ss_fac);
     fprintf(outfile,"        SCS-MP2 correlation energy:     %20.12lf\n",emp2_os*emp2_os_fac+emp2_ss*emp2_ss_fac);
     fprintf(outfile,"      * SCS-MP2 total energy:           %20.12lf\n",emp2_os*emp2_os_fac+emp2_ss*emp2_ss_fac+escf);
     fprintf(outfile,"\n");
  }
  fprintf(outfile,"        OS MP2 correlation energy:      %20.12lf\n",emp2_os);
  fprintf(outfile,"        SS MP2 correlation energy:      %20.12lf\n",emp2_ss);
  fprintf(outfile,"        MP2 correlation energy:         %20.12lf\n",emp2);
  fprintf(outfile,"      * MP2 total energy:               %20.12lf\n",emp2+escf);
  fprintf(outfile,"\n");
  if (options.get_bool("SCS_CCSD")){
     fprintf(outfile,"        OS SCS-CCSD correlation energy: %20.12lf\n",eccsd_os*eccsd_os_fac);
     fprintf(outfile,"        SS SCS-CCSD correlation energy: %20.12lf\n",eccsd_ss*eccsd_ss_fac);
     fprintf(outfile,"        SCS-CCSD correlation energy:    %20.12lf\n",eccsd_os*eccsd_os_fac+eccsd_ss*eccsd_ss_fac);
     fprintf(outfile,"      * SCS-CCSD total energy:          %20.12lf\n",eccsd_os*eccsd_os_fac+eccsd_ss*eccsd_ss_fac+escf);
     fprintf(outfile,"\n");
  }
  fprintf(outfile,"        OS CCSD correlation energy:     %20.12lf\n",eccsd_os);
  fprintf(outfile,"        SS CCSD correlation energy:     %20.12lf\n",eccsd_ss);
  fprintf(outfile,"        CCSD correlation energy:        %20.12lf\n",eccsd);
  fprintf(outfile,"      * CCSD total energy:              %20.12lf\n",eccsd+escf);
  fprintf(outfile,"\n");
  fprintf(outfile,"  Total time for CCSD iterations: %10.2lf s (user)\n",user_stop-user_start);
  fprintf(outfile,"                                  %10.2lf s (system)\n",sys_stop-sys_start);
  fprintf(outfile,"                                  %10d s (total)\n",(int)time_stop-(int)time_start);
  fprintf(outfile,"\n");
  fprintf(outfile,"  Time per iteration:             %10.2lf s (user)\n",(user_stop-user_start)/(iter-1));
  fprintf(outfile,"                                  %10.2lf s (system)\n",(sys_stop-sys_start)/(iter-1));
  fprintf(outfile,"                                  %10.2lf s (total)\n",((double)time_stop-(double)time_start)/(iter-1));

  fflush(stdout);
  fflush(outfile);

  // free some of the cpu memory before exiting in case we end up doing triples next
  hipHostFree(E2ijkl);
  hipHostFree(E2ijak2);
  hipHostFree(E2ijak3);
  hipHostFree(E2ijakCopy);
  hipHostFree(E2akjc_2);
  hipHostFree(Symabcd1);
  hipHostFree(Symabcd2);
  hipHostFree(tempu);
  hipHostFree(w1);
  hipHostFree(I1);
  hipHostFree(I1p);

  return Success;
}
void GPUCoupledCluster::SCS_CCSD(){
  long int v = nvirt;
  long int o = ndoccact;
  long int iajb,ijab=0;
  double ssenergy = 0.0;
  double osenergy = 0.0;
  for (long int a=o; a<nmo; a++){
      for (long int b=o; b<nmo; b++){
          for (long int i=0; i<o; i++){
              for (long int j=0; j<o; j++){

                  iajb = i*v*v*o+(a-o)*v*o+j*v+(b-o);
                  osenergy += E2klcd_1[iajb]*(tb[ijab]+t1[(a-o)*o+i]*t1[(b-o)*o+j]);
                  ssenergy += E2klcd_1[iajb]*(tb[ijab]-tb[(b-o)*o*o*v+(a-o)*o*o+i*o+j]);
                  ssenergy += E2klcd_1[iajb]*(t1[(a-o)*o+i]*t1[(b-o)*o+j]-t1[(b-o)*o+i]*t1[(a-o)*o+j]);
                  ijab++;
              }
          }
      }
  }
  eccsd_os = osenergy;
  eccsd_ss = ssenergy;
}
void GPUCoupledCluster::SCS_MP2(){
  long int v = nvirt;
  long int o = ndoccact;
  long int iajb,ijab=0;
  double ssenergy = 0.0;
  double osenergy = 0.0;
  for (long int a=o; a<nmo; a++){
      for (long int b=o; b<nmo; b++){
          for (long int i=0; i<o; i++){
              for (long int j=0; j<o; j++){
                  iajb = i*v*v*o+(a-o)*v*o+j*v+(b-o);
                  osenergy += E2klcd_1[iajb]*tb[ijab];
                  ssenergy += E2klcd_1[iajb]*(tb[ijab]-tb[(b-o)*o*o*v+(a-o)*o*o+i*o+j]);
                  ijab++;
              }
          }
      }
  }
  emp2_os = osenergy;
  emp2_ss = ssenergy;
}
double GPUCoupledCluster::CheckEnergy(){
  long int v = nvirt;
  long int o = ndoccact;
  long int iajb,jaib,ijab=0;
  double energy = 0.0;
  for (long int a=o; a<nmo; a++){
      for (long int b=o; b<nmo; b++){
          for (long int i=0; i<o; i++){
              for (long int j=0; j<o; j++){
                  iajb = i*v*v*o+(a-o)*v*o+j*v+(b-o);
                  jaib = iajb + (i-j)*v*(1-v*o);
                  energy += (2.*E2klcd_1[iajb]-E2klcd_1[jaib])*(tb[ijab]+t1[(a-o)*o+i]*t1[(b-o)*o+j]);
                  ijab++;
              }
          }
      }
  }
  return energy;
}
/*===================================================================

  allocate memory on gpu

===================================================================*/
void GPUCoupledCluster::AllocateGPUMemory(){
  int o2 = ndoccact*ndoccact;
  int ov = ndoccact*nvirt;
  int v2 = nvirt*nvirt;
  hipMalloc((void**)&gpuv,sizeof(double)*((left-wasted))); 
  Check_CUDA_Error(outfile,"gpuv");
  hipMalloc((void**)&gput2,sizeof(double)*o2*v2);      
  Check_CUDA_Error(outfile,"gput2");
  hipMalloc((void**)&gpuw,sizeof(double)*o2*v2);       
  Check_CUDA_Error(outfile,"gpuw");
  hipMalloc((void**)&gputempw,sizeof(double)*o2*v2);   
  Check_CUDA_Error(outfile,"gputempw");
  hipMalloc((void**)&gput1,sizeof(double)*ov);         
  Check_CUDA_Error(outfile,"gput1");
  hipMalloc((void**)&gpuw1,sizeof(double)*nmo*nmo);      
  Check_CUDA_Error(outfile,"gpuw1");
}
/*===================================================================

  free gpu memory and mapped cpu memory

===================================================================*/
void GPUCoupledCluster::CudaFinalize(){
  hipFree(gpuv);
  hipFree(gpuw);
  hipFree(gputempw);
  hipFree(gput2);
  hipFree(gput1);
  hipHostFree(E2ijak);
  hipHostFree(E2klcd_1);
  hipHostFree(E2abci);
  hipHostFree(tb);
  hipHostFree(t1);
  hipDeviceReset();
}
/*===================================================================

  determine tiling for vabcd and vabci diagrams

===================================================================*/
void GPUCoupledCluster::DefineTiling(){
  int i,v = nvirt;
  int o = ndoccact;
  int ov2 = o*v*v;
  int ov = o*v;
  int o2 = o*o;
  int h = v;
  wasted = 350*1024*1024/8.; // leave an extra 200 mb on there.
  ntiles = -999;

  // check whether blocking of the vabcd diagram is necessary
  if (left-wasted>v*(v+1)/2*v*(v+1)/2){
     tilesize = v*(v+1)/2;
     ntiles = 1;
  }
  else{
     for (i=2; i<=v*(v+1)/2; i++){
         if (left-wasted>(double)tilesize*v*(v+1)/2/i+1){
            tilesize = v*(v+1)/2/i;
            if (i*tilesize < v*(v+1)/2) tilesize++;
            ntiles = i;
            break;
         }
     }
     if (ntiles==-999){
        fprintf(outfile,"\n  error. Not enough device memory.\n\n");
        fflush(outfile);
        exit(0);
     }
  }
  lasttile = v*(v+1)/2 - (ntiles-1)*tilesize;
  if (tilesize<o2){
     fprintf(outfile,"\n  error. Not enough device memory. maybe.\n\n");
     fflush(outfile);
  }
  if (ntiles>1){
     fprintf(outfile,
       "  v(ab,cd) diagram will be evaluated in %5i blocks.\n\n",ntiles); 
     fflush(outfile);
  }
  nov2tiles=1;
  ov2tilesize=ov2/1;
  if (nov2tiles*ov2tilesize<ov2) ov2tilesize++;
  while(h*ov2tilesize>left-wasted){
     nov2tiles++;
     ov2tilesize = ov2/nov2tiles;
     if (nov2tiles*ov2tilesize<ov2) ov2tilesize++;
  }
  if (nov2tiles>1){
     fprintf(outfile,
       "  v(ab,ci) terms will be evaluated in   %5i blocks.\n\n",nov2tiles); 
     fflush(outfile);
  }
  lastov2tile = ov2 - (nov2tiles-1)*ov2tilesize;

  novtiles=1;
  ovtilesize=ov/1;
  if (novtiles*ovtilesize<ov) ovtilesize++;
  while(h*h*ovtilesize>left-wasted){
     novtiles++;
     novtiles++;
     ovtilesize = ov/novtiles;
     if (novtiles*ovtilesize<ov) ovtilesize++;
  }
  lastovtile = ov - (novtiles-1)*ovtilesize;
}
/*===================================================================

  initialize cublas and get device properties

===================================================================*/
void GPUCoupledCluster::CudaInit(){
  cublasInit();
  struct hipDeviceProp_t cudaProp;
  int gpu_id;
  hipGetDevice(&gpu_id);
  hipGetDeviceProperties( &cudaProp,gpu_id );
  fprintf(outfile,
    "\n  _________________________________________________________\n");
  fprintf(outfile,"  CUDA device properties:\n");
  fprintf(outfile,"  name:                 %20s\n",cudaProp.name);
  fprintf(outfile,"  major version:        %20d\n",cudaProp.major);
  fprintf(outfile,"  minor version:        %20d\n",cudaProp.minor);
  fprintf(outfile,"  canMapHostMemory:     %20d\n",cudaProp.canMapHostMemory);
  fprintf(outfile,"  totalGlobalMem:       %20lu mb\n",
    cudaProp.totalGlobalMem/(1024*1024));
  fprintf(outfile,"  sharedMemPerBlock:    %20lu\n",cudaProp.sharedMemPerBlock);
  fprintf(outfile,"  clockRate:            %20.3f ghz\n",
    cudaProp.clockRate/1.0e6);
  fprintf(outfile,"  regsPerBlock:         %20d\n",cudaProp.regsPerBlock);
  fprintf(outfile,"  warpSize:             %20d\n",cudaProp.warpSize);
  fprintf(outfile,"  maxThreadsPerBlock:   %20d\n",cudaProp.maxThreadsPerBlock);
  fprintf(outfile,
    "  _________________________________________________________\n\n");
  fflush(outfile);
  // device memory left after some arrays
  int o = ndoccact;
  int v = nvirt;
  left = cudaProp.totalGlobalMem/8. - 3*o*o*v*v - o*v-nmo*nmo;

  nthreads=NUMTHREADS;
  num=1;
  if ((o*o*v*v)%nthreads==0)
     nblocks = (o*o*v*v)/nthreads;
  else
     nblocks = (o*o*v*v+nthreads-(o*o*v*v)%nthreads)/nthreads;
  if (nblocks>MAXBLOCKS){
     num = nblocks/MAXBLOCKS+1;
     nblocks = nblocks/num + 1;
  }
}
/*===================================================================

  read integrals from disk

===================================================================*/
void GPUCoupledCluster::ReadIntegrals(){
  int v = nvirt;
  int o = ndoccact;
  int k;

  boost::shared_ptr<PSIO> psio(new PSIO());

  psio->open(PSIF_IJAK,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_IJAK,"E2ijak",(char*)&E2ijak[0],o*o*o*v*sizeof(double));
  psio->close(PSIF_IJAK,1);

  psio->open(PSIF_ABCI,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_ABCI,"E2abci",(char*)&E2abci[0],o*v*v*v*sizeof(double));
  psio->close(PSIF_ABCI,1);

  psio->open(PSIF_ABCD1,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_ABCD1,"E2abcd1",(char*)&Symabcd1[0],v*(v+1)/2*v*(v+1)/2*sizeof(double));
  psio->close(PSIF_ABCD1,1);

  psio->open(PSIF_ABCD2,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_ABCD2,"E2abcd2",(char*)&Symabcd2[0],v*(v+1)/2*v*(v+1)/2*sizeof(double));
  psio->close(PSIF_ABCD2,1);

  psio->open(PSIF_IJKL,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_IJKL,"E2ijkl",(char*)&E2ijkl[0],o*o*o*o*sizeof(double));
  psio->close(PSIF_IJKL,1);

  psio->open(PSIF_AKJC2,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_AKJC2,"E2akjc2",(char*)&E2akjc_2[0],o*o*v*v*sizeof(double));
  psio->close(PSIF_AKJC2,1);

  psio->open(PSIF_KLCD,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_KLCD,"E2klcd",(char*)&E2klcd_1[0],o*o*v*v*sizeof(double));
  psio->close(PSIF_KLCD,1);

  int i,j,a,id=0;
  for (i=0; i<o; i++){
  for (a=0; a<v; a++){
  for (j=0; j<o; j++){
  for (k=0; k<o; k++){
      E2ijakCopy[id] = E2ijak[id];
      E2ijak2[id++] = E2ijak[i*o*o*v+k*o*v+j*v+a];
  }}}}
  id=0;
  for (j=0; j<o; j++){
  for (i=0; i<o; i++){
  for (k=0; k<o; k++){
  for (a=0; a<v; a++){
      E2ijak3[id++] = E2ijak[i*o*o*v+j*o*v+k*v+a];
  }}}}

}
/*===================================================================

  sort and write integrals to disk

===================================================================*/
void GPUCoupledCluster::WriteIntegrals(double*tei){
  int i,j,k,l,a,b,c,d,e,m,f;
  double dum;

  boost::shared_ptr<PSIO> psio(new PSIO());
  psio_address addr;

  double*tmpei;
  long int dim = nvirt*(nvirt+1)/2;
  dim = dim*dim;
  tmpei = (double*)malloc(dim*sizeof(double));
  int count=0;

  // E<ij|ak>
  psio->open(PSIF_IJAK,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  for (j=nfzc; j<ndocc; j++){
      for (i=nfzc; i<ndocc; i++){
          for (k=nfzc; k<ndocc; k++){
              for (a=ndocc; a<nmotemp; a++){
                  dum = tei[Position(Position(i-nfzc,a-nfzc),Position(j-nfzc,k-nfzc))];
                  tmpei[count++] = dum;
                  if (count==dim){
                     psio->write(PSIF_IJAK,"E2ijak",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
                     count=0;
                  }
              }
          }
      }
  }
  if (count>0)
     psio->write(PSIF_IJAK,"E2ijak",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_IJAK,1);

  // E<ab|ci>
  psio->open(PSIF_ABCI,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (i=nfzc; i<ndocc; i++){
      for (a=ndocc; a<nmotemp; a++){
          for (b=ndocc; b<nmotemp; b++){
              for (c=ndocc; c<nmotemp; c++){
                  dum = tei[Position(Position(a-nfzc,c-nfzc),Position(b-nfzc,i-nfzc))];
                  tmpei[count++] = dum;
                  if (count==dim){
                     psio->write(PSIF_ABCI,"E2abci",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
                     count=0;
                  }
              }
          }
      }
  }
  if (count>0)
     psio->write(PSIF_ABCI,"E2abci",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_ABCI,1);

  // E<ab|cd>
  psio->open(PSIF_ABCD1,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (a=ndocc; a<nmotemp; a++){
  for (b=ndocc; b<=a; b++){
  for (c=ndocc; c<nmotemp; c++){
  for (d=ndocc; d<=c; d++){
      if (c==d) dum = tei[Position(Position(a-nfzc,c-nfzc),Position(b-nfzc,d-nfzc))];
      else      dum = tei[Position(Position(a-nfzc,c-nfzc),Position(b-nfzc,d-nfzc))]
                    + tei[Position(Position(a-nfzc,d-nfzc),Position(b-nfzc,c-nfzc))];
      tmpei[count++] = dum;
      if (count==dim){
         psio->write(PSIF_ABCD1,"E2abcd1",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
         count=0;
      }

  }}}}
  if (count>0)
     psio->write(PSIF_ABCD1,"E2abcd1",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_ABCD1,1);


  psio->open(PSIF_ABCD2,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (a=ndocc; a<nmotemp; a++){
  for (b=ndocc; b<=a; b++){
  for (c=ndocc; c<nmotemp; c++){
  for (d=ndocc; d<=c; d++){
      dum = tei[Position(Position(a-nfzc,c-nfzc),Position(b-nfzc,d-nfzc))]
          - tei[Position(Position(a-nfzc,d-nfzc),Position(b-nfzc,c-nfzc))];
      tmpei[count++] = dum;
      if (count==dim){
         psio->write(PSIF_ABCD2,"E2abcd2",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
         count=0;
      }
  }}}}
  if (count>0)
     psio->write(PSIF_ABCD2,"E2abcd2",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_ABCD2,1);


  // E<ij|kl>
  psio->open(PSIF_IJKL,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (k=nfzc; k<ndocc; k++){
      for (l=nfzc; l<ndocc; l++){
          for (i=nfzc; i<ndocc; i++){
              for (j=nfzc; j<ndocc; j++){
                  dum = tei[Position(Position(i-nfzc,k-nfzc),Position(j-nfzc,l-nfzc))];
                  tmpei[count++] = dum;
                  if (count==dim){
                     psio->write(PSIF_IJKL,"E2ijkl",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
                     count=0;
                  }
              }
          }
      }
  }
  if (count>0)
     psio->write(PSIF_IJKL,"E2ijkl",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_IJKL,1);

  // E<ak|jc>
  psio->open(PSIF_AKJC2,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (k=nfzc; k<ndocc; k++){
      for (c=ndocc; c<nmotemp; c++){
          for (j=nfzc; j<ndocc; j++){
              for (a=ndocc; a<nmotemp; a++){
                  dum = tei[Position(Position(a-nfzc,c-nfzc),Position(k-nfzc,j-nfzc))];
                  tmpei[count++] = dum;
                  if (count==dim){
                     psio->write(PSIF_AKJC2,"E2akjc2",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
                     count=0;
                  }
              }
          }
      }
  }
  if (count>0)
     psio->write(PSIF_AKJC2,"E2akjc2",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_AKJC2,1);

  // E<kl|cd>
  psio->open(PSIF_KLCD,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (k=nfzc; k<ndocc; k++){
      for (c=ndocc; c<nmotemp; c++){
          for (l=nfzc; l<ndocc; l++){
              for (d=ndocc; d<nmotemp; d++){
                  dum = tei[Position(Position(k-nfzc,c-nfzc),Position(l-nfzc,d-nfzc))];
                  tmpei[count++] = dum;
                  if (count==dim){
                     psio->write(PSIF_KLCD,"E2klcd",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
                     count=0;
                  }
              }
          }
      }
  }
  if (count>0)
     psio->write(PSIF_KLCD,"E2klcd",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_KLCD,1);

  // these won't get deleted:
  /*psio->open(PSIF_ABCI5,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (a=ndocc; a<nmotemp; a++){
  for (b=ndocc; b<nmotemp; b++){
  for (i=nfzc; i<ndocc; i++){
  for (c=ndocc; c<nmotemp; c++){
      dum = tei[Position(Position(a-nfzc,c-nfzc),Position(b-nfzc,i-nfzc))];
      tmpei[count++] = dum;
      if (count==ndoccact*ndoccact*nvirt*nvirt){
         psio->write(PSIF_ABCI5,"E2abci5",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
         count=0;
      }
      //psio->write(PSIF_ABCI5,"E2abci5",(char*)&dum,sizeof(double),addr,&addr);
  }}}}
  if (count>0)
     psio->write(PSIF_ABCI5,"E2abci5",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_ABCI5,1);*/

  psio->open(PSIF_ABCI2,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (a=ndocc; a<nmotemp; a++){
  for (b=ndocc; b<nmotemp; b++){
  for (e=ndocc; e<nmotemp; e++){
  for (m=nfzc; m<ndocc; m++){
      dum = 2.*tei[Position(Position(a-nfzc,b-nfzc),Position(e-nfzc,m-nfzc))]
          -    tei[Position(Position(a-nfzc,e-nfzc),Position(b-nfzc,m-nfzc))];
      tmpei[count++] = dum;
      if (count==dim){
         psio->write(PSIF_ABCI2,"E2abci2",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
         count=0;
      }
  }}}}
  if (count>0)
     psio->write(PSIF_ABCI2,"E2abci2",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_ABCI2,1);

  psio->open(PSIF_ABCI3,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (a=ndocc; a<nmotemp; a++){
  for (f=ndocc; f<nmotemp; f++){
  for (m=nfzc; m<ndocc; m++){
  for (e=ndocc; e<nmotemp; e++){
      dum = tei[Position(Position(a-nfzc,f-nfzc),Position(e-nfzc,m-nfzc))];
      tmpei[count++] = dum;
      if (count==dim){
         psio->write(PSIF_ABCI3,"E2abci3",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
         count=0;
      }
  }}}}
  if (count>0)
     psio->write(PSIF_ABCI3,"E2abci3",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_ABCI3,1);

  psio->open(PSIF_ABCI4,PSIO_OPEN_NEW);
  addr = PSIO_ZERO;
  count=0;
  for (i=nfzc; i<ndocc; i++){
  for (a=ndocc; a<nmotemp; a++){
  for (b=ndocc; b<nmotemp; b++){
  for (c=ndocc; c<nmotemp; c++){
      dum = tei[Position(Position(a-nfzc,b-nfzc),Position(c-nfzc,i-nfzc))];
      tmpei[count++] = dum;
      if (count==dim){
         psio->write(PSIF_ABCI4,"E2abci4",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
         count=0;
      }
  }}}}
  if (count>0)
     psio->write(PSIF_ABCI4,"E2abci4",(char*)&tmpei[0],count*sizeof(double),addr,&addr);
  psio->close(PSIF_ABCI4,1);

  psio.reset();
  free(tmpei);
}
/*===================================================================

  allocate cpu memory

===================================================================*/
void GPUCoupledCluster::AllocateMemory(){

  int o=ndoccact;
  int v=nvirt;
  int dim = v*(v+1)/2; 
  dim = dim*dim;

  // integrals:

  // o^4
  hipHostMalloc((void**)&E2ijkl,o*o*o*o*sizeof(double));      Check_CUDA_Error(outfile,"cpu E2ijkl");

  // o^3v
  hipHostMalloc((void**)&E2ijak,o*o*o*v*sizeof(double));      Check_CUDA_Error(outfile,"cpu E2ijak");
  hipHostMalloc((void**)&E2ijak2,o*o*o*v*sizeof(double));     Check_CUDA_Error(outfile,"cpu E2ijak2");
  hipHostMalloc((void**)&E2ijak3,o*o*o*v*sizeof(double));     Check_CUDA_Error(outfile,"cpu E2ijak3");
  hipHostMalloc((void**)&E2ijakCopy,o*o*o*v*sizeof(double));  Check_CUDA_Error(outfile,"cpu E2ijakCopy");

  // o^2v^2
  hipHostMalloc((void**)&E2akjc_2,o*o*v*v*sizeof(double));    Check_CUDA_Error(outfile,"cpu E2akjc_2");
  hipHostMalloc((void**)&E2klcd_1,o*o*v*v*sizeof(double));    Check_CUDA_Error(outfile,"cpu E2klcd_1");

  // ov^3
  hipHostMalloc((void**)&E2abci,o*v*v*v*sizeof(double));      Check_CUDA_Error(outfile,"cpu E2abci");

  // v(v+1)v(v+1)/4
  hipHostMalloc((void**)&Symabcd1,dim*sizeof(double));        Check_CUDA_Error(outfile,"cpu Symabcd1");
  hipHostMalloc((void**)&Symabcd2,dim*sizeof(double));        Check_CUDA_Error(outfile,"cpu Symabcd2");

  // extra buffers:
  hipHostMalloc((void**)&tempu,o*o*v*v*sizeof(double));  Check_CUDA_Error(outfile,"cpu tempu");
  hipHostMalloc((void**)&tb,o*o*v*v*sizeof(double));     Check_CUDA_Error(outfile,"cpu tb");
  hipHostMalloc((void**)&w1,o*v*sizeof(double));         Check_CUDA_Error(outfile,"cpu w1");
  hipHostMalloc((void**)&I1,v*v*sizeof(double));         Check_CUDA_Error(outfile,"cpu I1");
  hipHostMalloc((void**)&I1p,v*v*sizeof(double));        Check_CUDA_Error(outfile,"cpu I1p");
  hipHostMalloc((void**)&t1,o*v*sizeof(double));         Check_CUDA_Error(outfile,"cpu t1");

  // these don't need to be pinned and will never be touched by a gpu function
  wb    = (double*)malloc(sizeof(double)*o*o*v*v);
  tempt = (double*)malloc(sizeof(double)*(o*o*v*v+o*v));
  tempv = (double*)malloc(sizeof(double)*(o*o*v*v+o*v));

  memset((void*)E2ijak,'\0',o*o*o*v*sizeof(double));
  memset((void*)E2ijak2,'\0',o*o*o*v*sizeof(double));
  memset((void*)E2ijak3,'\0',o*o*o*v*sizeof(double));
  memset((void*)E2ijakCopy,'\0',o*o*o*v*sizeof(double));
  memset((void*)E2ijkl,'\0',o*o*o*o*sizeof(double));
  memset((void*)E2abci,'\0',o*v*v*v*sizeof(double));
  memset((void*)tempu,'\0',o*o*v*v*sizeof(double));
  memset((void*)wb,'\0',o*o*v*v*sizeof(double));
  memset((void*)tb,'\0',o*o*v*v*sizeof(double));
  memset((void*)E2klcd_1,'\0',o*o*v*v*sizeof(double));
  memset((void*)E2akjc_2,'\0',o*o*v*v*sizeof(double));
  memset((void*)w1,'\0',o*v*sizeof(double));
  memset((void*)t1,'\0',o*v*sizeof(double));
  memset((void*)I1,'\0',v*v*sizeof(double));
  memset((void*)I1p,'\0',v*v*sizeof(double));

  // DIIS:
  diisvec    = (double*)malloc(sizeof(double)*(maxdiis+1));
  memset((void*)diisvec,'\0',(maxdiis+1)*sizeof(double));
}

void GPUCoupledCluster::CPU_t1_vmeai(){
  int o=ndoccact;
  int v=nvirt;
  for (int e=0; e<v; e++){
      for (int m=0; m<o; m++){
          for (int i=0; i<o; i++){
              F_DCOPY(v,E2akjc_2+m*o*v*v+e*o*v+i*v,1,tempv+e*o*o*v+m*o*v+i,o);
              F_DAXPY(v,-2.0,E2klcd_1+m*o*v*v+e*o*v+i*v,1,tempv+e*o*o*v+m*o*v+i,o);
          }
      }
  }
  F_DGEMV('n',o*v,o*v,-1.0,tempv,o*v,t1,1,1.0,w1,1);

}

void GPUCoupledCluster::CPU_t1_vmeni(){
  int o=ndoccact;
  int v=nvirt;
  for (int a=0; a<v; a++){
      for (int m=0; m<o; m++){
          for (int n=0; n<o; n++){
              F_DCOPY(v,tb+a*v*o*o+m*o+n,o*o,tempt+a*o*o*v+m*o*v+n*v,1);
              F_DAXPY(v,-2.0,tb+a*o*o+m*o+n,o*o*v,tempt+a*o*o*v+m*o*v+n*v,1);
          }
      }
  }
  F_DGEMM('t','n',o,v,o*o*v,1.0,E2ijak,o*o*v,tempt,o*o*v,1.0,w1,o);
}

void GPUCoupledCluster::CPU_t1_vmaef(){
  int m,e,i,f;
  int o=ndoccact;
  int v=nvirt;
  for (f=0; f<v; f++){
      for (m=0; m<o; m++){
          for (e=0; e<v; e++){
              F_DCOPY(o,tb+e*v*o*o+f*o*o+m,o,tempt+f*o*o*v+m*o*v+e*o,1);
              F_DAXPY(o,-2.0,tb+e*v*o*o+f*o*o+m*o,1,tempt+f*o*o*v+m*o*v+e*o,1);
          }
      }
  }

  int tilesize,lasttile,ntiles=1;
  int ov2 = o*v*v;
  // tile v in chunks of o
  tilesize=v;
  for (i=1; i<=v; i++){
      if (o>=(double)tilesize/i){
         tilesize = v/i;
         if (i*tilesize < v) tilesize++;
         ntiles = i;
         break;
      }
  }
  lasttile = v - (ntiles-1)*tilesize;
  boost::shared_ptr<PSIO> psio(new PSIO());
  psio->open(PSIF_ABCI3,PSIO_OPEN_OLD);
  psio_address addr;
  addr = PSIO_ZERO;
  for (i=0; i<ntiles-1; i++){
      psio->read(PSIF_ABCI3,"E2abci3",(char*)&tempv[0],tilesize*ov2*sizeof(double),addr,&addr);
      F_DGEMM('n','n',o,tilesize,ov2,-1.0,tempt,o,tempv,ov2,1.0,w1+i*tilesize*o,o);
  }
  i=ntiles-1;
  psio->read(PSIF_ABCI3,"E2abci3",(char*)&tempv[0],lasttile*ov2*sizeof(double),addr,&addr);
  F_DGEMM('n','n',o,lasttile,ov2,-1.0,tempt,o,tempv,ov2,1.0,w1+i*tilesize*o,o);
  psio->close(PSIF_ABCI3,1);
  psio.reset();
}

void GPUCoupledCluster::CPU_I1ab(){
  int b,m,n,e,a,id;
  int o = ndoccact;
  int v = nvirt;
  // build I1(a,b)
  for (m=0; m<o; m++){
      for (e=0; e<v; e++){
          for (n=0; n<o; n++){
              F_DCOPY(v,E2klcd_1+m*o*v*v+n*v+e,o*v,tempv+m*o*v*v+e*o*v+n*v,1);
          }
      }
  }
  F_DAXPY(o*o*v*v,-2.0,E2klcd_1,1,tempv,1);

  for (m=0,id=0; m<o; m++){
      for (e=0; e<v; e++){
          for (n=0; n<o; n++){
              F_DCOPY(v,tb+e*v*o*o+m*o+n,o*o,tempt+m*o*v*v+e*o*v+n*v,1);
              for (b=0; b<v; b++){
                  tempt[id++] += t1[e*o+m]*t1[b*o+n];
              }
          }
      }
  }
  F_DGEMM('n','t',v,v,o*o*v,1.0,tempv,v,tempt,v,0.0,I1,v);

  // add the singles parts to I1(a,b). n^4
  // TODO: can tile in larger blocks
  boost::shared_ptr<PSIO> psio(new PSIO());
  psio->open(PSIF_ABCI2,PSIO_OPEN_OLD);
  psio_address addr = PSIO_ZERO;
  for (a=0; a<v; a++){
      psio->read(PSIF_ABCI2,"E2abci2",(char*)&tempt[0],v*v*o*sizeof(double),addr,&addr);
      F_DGEMV('t',o*v,v,1.0,tempt,o*v,t1,1,1.0,I1+a*v,1);
  }
  psio->close(PSIF_ABCI2,1);
  psio.reset();

  int l,k,c;
  for (l=0; l<o; l++){
      for (c=0; c<v; c++){
          for (k=0; k<o; k++){
              F_DCOPY(v,tb+c*o*o+l*o+k,o*o*v,tempt+l*o*v*v+c*o*v+k*v,1);
          }
      }
  }
  // use I1(a,b) for doubles residual:
  F_DGEMM('t','n',v,o*o*v,v,1.0,I1,v,tempt,v,0.0,tempv,v);
  for (int a=0; a<v; a++){
      for (int b=0; b<v; b++){
          for (int i=0; i<o; i++){
              F_DAXPY(o,1.0,tempv+a*v*o+i*v+b,v*v*o,wb+a*o*o*v+b*o*o+i*o,1);
              F_DAXPY(o,1.0,tempv+i*v*v*o+b*v*o+a,v,wb+a*o*o*v+b*o*o+i*o,1);
          }
      }
  }

  // use I1(a,b) for singles residual - 1st contribution to w1. (n^3)
  F_DGEMM('n','n',o,v,v,1.0,t1,o,I1,v,1.0,w1,o);
}


// CPU_I2p_abci required ov^3 storage.  by refactorizing, we reduce storage to o^3v, but increase cost by 2o^2v^3
// who cares, this sits on the cpu anyway
// TODO: move terms around like in plugin_ccsd_serial
void GPUCoupledCluster::CPU_I2p_abci_refactored(){
  int a,b,i;
  int o = ndoccact;
  int v = nvirt;
  int ov2 = o*v*v;
  int o2v = o*o*v;

  // tilesize * v <= o^2v^2
  int tilesize,lasttile,ntiles=1;
  tilesize=ov2;
  for (i=1; i<=ov2; i++){
      if (o*o*v*v>=(double)tilesize*v/i){
         tilesize = ov2/i;
         if (i*tilesize < ov2) tilesize++;
         ntiles = i;
         break;
      }
  }
  lasttile = ov2 - (ntiles-1)*tilesize;
  boost::shared_ptr<PSIO> psio(new PSIO());
  psio->open(PSIF_ABCI,PSIO_OPEN_OLD);
  psio_address addr;
  addr = PSIO_ZERO;
  for (i=0; i<ntiles-1; i++){
      psio->read(PSIF_ABCI,"E2abci",(char*)&tempv[0],v*tilesize*sizeof(double),addr,&addr);
      F_DGEMM('n','n',o,tilesize,v,1.0,t1,o,tempv,v,0.0,tempt+i*tilesize*o,o);
  }
  i=ntiles-1;
  psio->read(PSIF_ABCI,"E2abci",(char*)&tempv[0],v*lasttile*sizeof(double),addr,&addr);
  F_DGEMM('n','n',o,lasttile,v,1.0,t1,o,tempv,v,0.0,tempt+i*tilesize*o,o);
  psio->close(PSIF_ABCI,1);
  psio.reset();
  
  for (a=0; a<v; a++){
      for (b=0; b<v; b++){
          for (i=0; i<o; i++){
              F_DAXPY(o,1.0,tempt+a*o*v+b*o+i,v*v*o,wb+a*o*o*v+b*o*o+i*o,1);
              F_DAXPY(o,1.0,tempt+i*v*v*o+b*o*v+a*o,1,wb+a*o*o*v+b*o*o+i*o,1);
          }
      }
  }


  // now build and use 2 new intermediates:
  F_DGEMM('n','n',o,o2v,v,-1.0,t1,o,E2akjc_2,v,0.0,tempt,o);
  F_DGEMM('n','n',o2v,v,o,1.0,tempt,o2v,t1,o,0.0,tempv,o2v);
  for (a=0; a<v; a++){
      for (b=0; b<v; b++){
          for (i=0; i<o; i++){
              F_DAXPY(o,1.0,tempv+b*v*o*o+a*o*o+i,o,wb+a*o*o*v+b*o*o+i*o,1);
              F_DAXPY(o,1.0,tempv+a*v*o*o+b*o*o+i*o,1,wb+a*o*o*v+b*o*o+i*o,1);
          }
      }
  }
  F_DGEMM('t','t',o2v,o,v,-1.0,E2klcd_1,v,t1,o,0.0,tempt,o2v);
  F_DGEMM('t','n',v,o2v,o,1.0,t1,o,tempt,o,0.0,tempv,v);
  for (a=0; a<v; a++){
      for (b=0; b<v; b++){
          for (i=0; i<o; i++){
              F_DAXPY(o,1.0,tempv+i*v*v*o+b*v+a,v*v,wb+a*o*o*v+b*o*o+i*o,1);
              F_DAXPY(o,1.0,tempv+i*v*v+a*v+b,o*v*v,wb+a*o*o*v+b*o*o+i*o,1);
          }
      }
  }
}

void GPUCoupledCluster::CPU_I1pij_I1ia_lessmem(){
  int m,j,e,i,a,b;
  int o = ndoccact;
  int v = nvirt;
  int ov2 = o*v*v;
  double*tempw1;
  tempw1 = (double*)malloc(o*v*sizeof(double));

  // build I1(i,a). n^4
  for (m=0; m<o; m++){
      for (e=0; e<v; e++){
          for (j=0; j<o; j++){
              F_DCOPY(v,tb+e*v*o*o+j*o+m,o*o,tempt+m*o*v*v+e*o*v+j*v,1);
              F_DAXPY(v,-2.0,tb+e*v*o*o+m*o+j,o*o,tempt+m*o*v*v+e*o*v+j*v,1);
          }
      }
  }
  for (i=0; i<o; i++){
      for (a=0; a<v; a++){
          for (m=0; m<o; m++){
              F_DCOPY(v,E2klcd_1+i*o*v*v+m*v+a,o*v,tempv+i*v*v*o+a*v*o+m,o);
              F_DAXPY(v,-2.0,E2klcd_1+i*o*v*v+a*o*v+m*v,1,tempv+i*v*v*o+a*v*o+m,o);
          }
      }
  }
  F_DGEMM('t','n',o*v,1,o*v,-1.0,tempv,o*v,t1,o*v,0.0,I1,o*v);

  // use I1(i,a) -> w1
  F_DGEMM('n','n',o*v,1,o*v,-1.0,tempt,o*v,I1,o*v,0.0,tempw1,o*v);
  for (i=0; i<o; i++){
      F_DAXPY(v,1.0,tempw1+i*v,1,w1+i,o);
  }

  // build I1'(i,j)
  F_DGEMM('t','n',o,o,ov2,-1.0,tempt,ov2,E2klcd_1,ov2,0.0,I1p,o);
  
  // only n^4
  for (i=0; i<o; i++){
      for (j=0; j<o; j++){
          for (m=0; m<o; m++){
              F_DCOPY(v,E2ijak+m*o*o*v+i*o*v+j*v,1,tempv+i*o*o*v+j*o*v+m,o);
              F_DAXPY(v,-2.0,E2ijak+i*o*o*v+m*o*v+j*v,1,tempv+i*o*o*v+j*o*v+m,o);
          }
      }
  }
  F_DGEMM('t','n',o*o,1,o*v,-1.0,tempv,o*v,t1,o*v,1.0,I1p,o*o);

  // use I1'(i,j) for singles residual. (n^3)
  F_DGEMM('n','n',o,v,o,-1.0,I1p,o,t1,o,1.0,w1,o);

  // build I1(i,j)
  F_DGEMM('n','n',o,o,v,1.0,t1,o,I1,v,1.0,I1p,o);
  for (m=0; m<o; m++){
      for (e=0; e<v; e++){
          for (j=0; j<o; j++){
              F_DCOPY(v,tb+e*o*o*v+m*o+j,o*o,tempt+m*o*v*v+e*o*v+j*v,1);
          }
      }
  }
  F_DGEMM('n','t',o,ov2,o,-1.0,I1p,o,tempt,ov2,0.0,tempv,o);
  for (a=0; a<v; a++){
      for (b=0; b<v; b++){
          for (i=0; i<o; i++){
              F_DAXPY(o,1.0,tempv+a*o*o*v+b*o+i,v*o,wb+a*o*o*v+b*o*o+i*o,1);
              F_DAXPY(o,1.0,tempv+b*o*o*v+i*v*o+a*o,1,wb+a*o*o*v+b*o*o+i*o,1);
          }
      }
  }
  free(tempw1);
}

/*================================================================

   update amplitudes

================================================================*/
void GPUCoupledCluster::UpdateT1(int iter){
  int v = nvirt;
  int o = ndoccact;
  int i,a;
  double tnew,dia;
  for (a=o; a<nmo; a++){
      for (i=0; i<o; i++){
          dia = -eps[i]+eps[a];
          tnew = - (w1[(a-o)*o+i])/dia;
          w1[(a-o)*o+i] = tnew;
      }
  }
  // error vector for diis is in tempv:
  F_DCOPY(o*v,w1,1,tempv+o*o*v*v,1);
  F_DAXPY(o*v,-1.0,t1,1,tempv+o*o*v*v,1);
  F_DCOPY(o*v,w1,1,t1,1);
}
void GPUCoupledCluster::UpdateT2(int iter){
  int v = nvirt;
  int o = ndoccact;
  double tnew,dijab,da,dab,dabi;
  int iajb,jaib,ijab=-1;
  for (long int a=o; a<nmo; a++){
      da = eps[a];
      for (long int b=o; b<nmo; b++){
          dab = da + eps[b];
          for (long int i=0; i<o; i++){
              dabi = dab - eps[i];
              for (long int j=0; j<o; j++){
                  ijab++;
                  iajb = i*v*v*o+(a-o)*v*o+j*v+(b-o);
                  jaib = iajb + (i-j)*v*(1-v*o);

                  dijab = dabi-eps[j];

                  tnew = - (E2klcd_1[iajb] + wb[ijab])/dijab;
                  tempt[ijab] = tnew;
                  tempu[ijab] = (2.*E2klcd_1[iajb]-E2klcd_1[jaib])*(tnew+t1[(a-o)*o+i]*t1[(b-o)*o+j]);
              }
          }
      }
  }
  F_DCOPY(o*o*v*v,tb,1,tempv,1);
  F_DAXPY(o*o*v*v,-1.0,tempt,1,tempv,1);
  F_DCOPY(o*o*v*v,tempt,1,tb,1);
}

/*================================================================

   gpu functions

================================================================*/

__global__ void AddPermutedOnGPU(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  out[id] = in[id] + in[b*h*ns*ns+a*ns*ns+j*ns+i];
}
__global__ void GPUFill_I2iajk_and_c2Sym1(int ns,int h,double*gpuv,double*gpuw,double*gput2,double*gputempw){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  gpuw[id] += gpuv[id] + gpuv[b*ns*ns*h+a*ns*ns+j*ns+i];
  if (i<=j && a<=b)
     gputempw[GPUPosition(a,b)*ns*(ns+1)/2+GPUPosition(i,j)] = gput2[id]+gput2[b*ns*ns*h+a*ns*ns+i*ns+j];
}
__global__ void GPUFill_I2iajk_add(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  out[id] += in[id] + in[b*ns*ns*h+a*ns*ns+j*ns+i];
}
__global__ void GPUc2Sym1(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  if (i>j) return;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  if (a>b) return;
  out[GPUPosition(a,b)*ns*(ns+1)/2+GPUPosition(i,j)] = in[id]+in[b*ns*ns*h+a*ns*ns+i*ns+j];
}
__global__ void GPUc2Sym2(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  if (i>j) return;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  if (a>b) return;
  out[GPUPosition(a,b)*ns*(ns+1)/2+GPUPosition(i,j)] = in[id]-in[b*ns*ns*h+a*ns*ns+i*ns+j];
}
__global__ void GPUSymmAdd2(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  short sg;
  short sg2=1;
  if (a>b) sg2 = -1;
  if (i>j) sg  = -sg2;
  else     sg  =  sg2;
  out[id] += sg*in[GPUPosition(a,b)*ns*(ns+1)/2+GPUPosition(i,j)];
}
__global__ void GPUSymmAdd1(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  out[id] += in[GPUPosition(a,b)*ns*(ns+1)/2+GPUPosition(i,j)];
}
__global__ void gpuCopyArray(double*out,double*in,int dim){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=dim) return;
  out[id] = in[id];
}

__global__ void  GPUc2Sym2_onefunction(int ns,int h,double*gput2,double*gput1,double*gputempw){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  if (i>j) return;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  if (a>b) return;
  gputempw[GPUPosition(a,b)*ns*(ns+1)/2+GPUPosition(i,j)] =
         gput2[id] - gput2[b*ns*ns*h+a*ns*ns+i*ns+j]
       + gput1[a*ns+i]*gput1[b*ns+j] - gput1[b*ns+i]*gput1[a*ns+j];
}

__global__ void GPUt2Plust1_and_E2klcd3(int ns,int h,double*t2,double*t1,double*E2klcd1,double*E2klcd3){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return; 
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  t2[id] += t1[a*ns+i]*t1[b*ns+j];

  E2klcd3[j*ns*h*h+i*h*h+b*h+a] = E2klcd1[j*ns*h*h+b*ns*h+i*h+a];
}
__global__ void GPUt2Plust1(int ns,int h,double*t2,double*t1){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return; 
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  t2[id] += t1[a*ns+i]*t1[b*ns+j];
}
__device__ int GPUPosition(int i,int j){
  if (i<j){
    return j*(j+1)/2+i;
  }
  return i*(i+1)/2+j;
}
__global__ void GPUt2Plus2t1(int ns,int h,double*out,double*t2,double*t1,double*newt2){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short a = id%h;
  unsigned short j = (id-a)%(ns*h)/h;
  unsigned short b = (id-a-j*h)%(ns*h*h)/(ns*h);
  unsigned short i = (id-a-j*h-b*ns*h)/(ns*h*h);
  //out[id] = t2[i*h*h*ns+a*h*ns+j*h+b] + 2.*t1[a*ns+i]*t1[b*ns+j];
  out[id]  = newt2[i*h*h*ns+a*h*ns+j*h+b] = t2[b*h*ns*ns+a*ns*ns+j*ns+i];
  out[id] += 2.*t1[a*ns+i]*t1[b*ns+j];
}
__global__ void GPUv2MinusHalfv2(int ns,int h,double*out,double*v2){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short a = id%h;
  unsigned short j = (id-a)%(ns*h)/h;
  unsigned short b = (id-a-j*h)%(ns*h*h)/(ns*h);
  unsigned short i = (id-a-j*h-b*ns*h)/(ns*h*h);
  out[id] = (v2[id]-.5*v2[i*h*h*ns+a*h*ns+j*h+b]);
}
__global__ void GPUPermute_iabj_to_aijb(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short b = (id-j)%(h*ns)/ns;
  unsigned short a = (id-j-b*ns)%(ns*h*h)/(h*ns);
  unsigned short i = (id-j-b*ns-a*h*ns)/(ns*h*h);
  out[a*ns*ns*h+i*ns*h+j*h+b] = in[id];
}
__global__ void GPUFill_I2iabj(int ns,int h,double*in1,double*in2,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short a = id%h;
  unsigned short j = (id-a)%(h*ns)/h;
  unsigned short b = (id-a-j*h)%(ns*h*h)/(h*ns);
  unsigned short i = (id-a-j*h-b*h*ns)/(ns*h*h);
  out[id] = in1[id] + in2[a*ns*ns*h+i*ns*h+j*h+b];
}
__global__ void GPU2t2Minust2(int ns,int h,double*out,double*t2){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short e = id%h;
  unsigned short m = (id-e)%(ns*h)/h;
  unsigned short a = (id-e-m*h)%(ns*h*h)/(ns*h);
  unsigned short i = (id-e-m*h-a*ns*h)/(ns*h*h);
  out[id] = 2.*t2[id]-t2[i*ns*h*h+e*ns*h+m*h+a];
}
__global__ void GPUFill_t2_I2iajb(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  out[id] = in[j*ns*h*h+b*h*ns+i*h+a] + in[i*ns*h*h+a*h*ns+j*h+b];
}
__global__ void GPUt2Plus2t1_and_E2klcd2(int ns,int h,double*out,double*t2,double*t1,double*E2klcd1,double*E2klcd2){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short a = id%h;
  unsigned short j = (id-a)%(ns*h)/h;
  unsigned short b = (id-a-j*h)%(ns*h*h)/(ns*h);
  unsigned short i = (id-a-j*h-b*ns*h)/(ns*h*h);
  out[id] = t2[j*h*h*ns+b*h*ns+i*h+a] + 2.*t1[a*ns+i]*t1[b*ns+j];
  //out[id] = t2[a*h*ns*ns+b*ns*ns+i*ns+j] + 2.*t1[a*ns+i]*t1[b*ns+j];

  E2klcd2[id] = E2klcd1[i*h*h*ns+a*h*ns+j*h+b]; 
}
__global__ void GPUt2Plus2t1_2(int ns,int h,double*out,double*t2,double*t1){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short a = id%h;
  unsigned short j = (id-a)%(ns*h)/h;
  unsigned short b = (id-a-j*h)%(ns*h*h)/(ns*h);
  unsigned short i = (id-a-j*h-b*ns*h)/(ns*h*h);
  out[id] = t2[j*h*h*ns+b*h*ns+i*h+a] + 2.*t1[a*ns+i]*t1[b*ns+j];
}
__global__ void GPUFillI2iajb2(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short a = id%h;
  unsigned short j = (id-a)%(h*ns)/h;
  unsigned short b = (id-a-j*h)%(ns*h*h)/(h*ns);
  unsigned short i = (id-a-j*h-b*h*ns)/(ns*h*h);
  out[id] += in[a*ns*ns*h+i*ns*h+j*h+b];
}
__global__ void GPUFillI2iajb1(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short a = id%h;
  unsigned short j = (id-a)%(h*ns)/h;
  unsigned short b = (id-a-j*h)%(ns*h*h)/(h*ns);
  unsigned short i = (id-a-j*h-b*h*ns)/(ns*h*h);
  out[id] += in[i*ns*h*h+a*h*ns+b*ns+j];
}
__global__ void GPUFill_t2_I2iabj1(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short j = id%ns;
  unsigned short i = (id-j)%(ns*ns)/ns;
  unsigned short b = (id-j-i*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-j-i*ns-b*ns*ns)/(ns*ns*h);
  out[id] = in[j*ns*h*h+b*h*ns+i*h+a] + in[i*ns*h*h+a*h*ns+j*h+b];
}
__global__ void GPUFill_t2_I2iabj2(int ns,int h,double*in,double*out){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short i = id%ns;
  unsigned short j = (id-i)%(ns*ns)/ns;
  unsigned short b = (id-i-j*ns)%(ns*ns*h)/(ns*ns);
  unsigned short a = (id-i-j*ns-b*ns*ns)/(ns*ns*h);
  out[id] += in[j*ns*h*h+b*h*ns+i*h+a] + in[i*ns*h*h+a*h*ns+j*h+b];
}
__global__ void GPUPermute_tikbc(double*in,double*out,int ns,int h){
  int blockid = blockIdx.x*gridDim.y + blockIdx.y;
  int id = blockid*blockDim.x + threadIdx.x;
  if (id>=ns*ns*h*h) return;
  unsigned short d = id%h;
  unsigned short k = (id-d)%(ns*h)/h;
  unsigned short c = (id-d-k*h)%(ns*h*h)/(ns*h);
  unsigned short l = (id-d-k*h-c*ns*h)/(ns*h*h);
  out[id] = in[l*h*h*ns+d*h*ns+k*h+c];
}

/*
 * truncated cepa
 */
void GPUCoupledCluster::TCEPA(){
  long int v = nvirt;
  long int o = ndoccact;
  long int rs = nmo;
  long int i,j,a,b;
  long int iajb,jaib,ijab=0;
  double energy = 0.0;
  memset((void*)tb,'\0',o*o*v*v*sizeof(double));
  double*pair_energy = (double*)malloc(o*o*sizeof(double));

  double tconv = 1.0;
  double di,dij,dija,dijab;
  while (tconv>1e-6){
      tconv = 0.0;
      for (i=0; i<o; i++){
          di = -eps[i];
          for (j=0; j<o; j++){
              dij = di - eps[j];
              energy=0.0;
              for (a=o; a<rs; a++){
                  for (b=o; b<rs; b++){
                      ijab = (a-o)*o*o*v+(b-o)*o*o+i*o+j;
                      iajb = i*v*v*o+(a-o)*v*o+j*v+(b-o);
                      jaib = j*v*v*o+(a-o)*v*o+i*v+(b-o);
                      energy += E2klcd_1[iajb]*(2.0*tb[ijab]-tb[(b-o)*o*o*v+(a-o)*o*o+i*o+j]);
                      //energy += (2.*E2klcd_1[iajb]-E2klcd_1[jaib])*(tb[ijab]+t1[(a-o)*o+i]*t1[(b-o)*o+j]);
                  }
              }
              pair_energy[i*o+j] = energy;
              for (a=o; a<rs; a++){
                  dija = dij + eps[a];
                  for (b=o; b<rs; b++){
                      dijab = dija + eps[b];
                      ijab = (a-o)*o*o*v+(b-o)*o*o+i*o+j;
                      iajb = i*v*v*o+(a-o)*v*o+j*v+(b-o);
                      double t2  = -E2klcd_1[iajb]/(dijab - pair_energy[i*o+j]);
                      double dum = (tb[ijab]-t2);
                      tconv += dum*dum;
                      tb[ijab] = t2;
                  }
              }
          }
      }
      for (a=0; a<v; a++){
          t1[a*o+i] = sqrt(fabs(tb[a*o*o*v+a*o*o+i*o+i]));
      }
      energy=0.0;
      for (i=0; i<o*o; i++) energy += pair_energy[i];
      printf("%20.12lf\n",energy);fflush(stdout);
      tconv = sqrt(tconv);
  }
  free(pair_energy);
}


}//end of namespace psi
