#include <libplugin/plugin.h>
#include"psi4-dec.h"
#include<boost/shared_ptr.hpp>
#include<liboptions/liboptions.h>
#include<libtrans/integraltransform.h>
#include<libtrans/mospace.h>
#include<libmints/matrix.h>
#include<libmints/vector.h>
#include<libchkpt/chkpt.h>
#include<libiwl/iwl.h>
#include <libpsio/psio.hpp>
#ifdef _OPENMP
  #include<omp.h>
#endif

// cuda libraries
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hip/hip_runtime.h>

#include"blas.h"
#include"gpuhelper.h"

using namespace psi;
using namespace boost;

namespace psi{

inline void GPUHelper::Check_CUDA_Error(FILE*fp,const char *message){
  hipError_t error = hipGetLastError();
  if (error!=hipSuccess) {
     fprintf(fp,"\n  ERROR: %s: %s\n\n", message, hipGetErrorString(error) );
     fflush(fp);
     exit(-1);
  }
}

/*===================================================================

  initialize cublas and get device properties

===================================================================*/
void GPUHelper::CudaInitGPU(Options&options){

  max_mapped_memory=0;
  num_gpus=gpumemory=extraroom=0;
  int n;
  hipGetDeviceCount(&n);
  num_gpus = n;
  num_cpus=1;
  if (num_gpus>0){
     cublasInit();
     struct hipDeviceProp_t cudaProp;
     int gpu_id;
     hipGetDevice(&gpu_id);
     hipGetDeviceProperties( &cudaProp,gpu_id );
     fprintf(outfile,
       "\n  _________________________________________________________\n");
     fprintf(outfile,"  CUDA device properties:\n");
     fprintf(outfile,"  name:                 %20s\n",cudaProp.name);
     fprintf(outfile,"  major version:        %20d\n",cudaProp.major);
     fprintf(outfile,"  minor version:        %20d\n",cudaProp.minor);
     fprintf(outfile,"  canMapHostMemory:     %20d\n",cudaProp.canMapHostMemory);
     fprintf(outfile,"  totalGlobalMem:       %20lu mb\n",
       cudaProp.totalGlobalMem/(1024*1024));
     fprintf(outfile,"  sharedMemPerBlock:    %20lu\n",cudaProp.sharedMemPerBlock);
     fprintf(outfile,"  clockRate:            %20.3f ghz\n",
       cudaProp.clockRate/1.0e6);
     fprintf(outfile,"  regsPerBlock:         %20d\n",cudaProp.regsPerBlock);
     fprintf(outfile,"  warpSize:             %20d\n",cudaProp.warpSize);
     fprintf(outfile,"  maxThreadsPerBlock:   %20d\n",cudaProp.maxThreadsPerBlock);
     fprintf(outfile,
       "  _________________________________________________________\n\n");
     fflush(outfile);

     gpumemory = cudaProp.totalGlobalMem;

     extraroom = 350L*1024L*1024L;
     
     hipDeviceReset();

     // default memory for mapped cpu memory is the sum of all gpu memory
     max_mapped_memory = (num_gpus+num_cpus) * (gpumemory-extraroom);
     if (options["MAX_MAPPED_MEMORY"].has_changed()){
        long int temp_mem = options.get_int("MAX_MAPPED_MEMORY");
        temp_mem *= 1024L*1024L;
        if (temp_mem<max_mapped_memory)
           max_mapped_memory = temp_mem;
     }
     max_mapped_memory_per_thread = max_mapped_memory/(num_gpus+num_cpus);

     fprintf(outfile,"\n");
     fprintf(outfile,"  allocating gpu memory...");
     fflush(outfile);
     tmp = (double**)malloc(num_gpus*sizeof(double*));
     gpuarray = (double**)malloc(num_gpus*sizeof(double*));
     #pragma omp parallel for schedule (static) num_threads(num_gpus)
     for (long int i=0; i<num_gpus; i++){
         long int thread = 0;
         #ifdef _OPENMP
           thread = omp_get_thread_num();
         #endif
         hipSetDevice(thread);
         Check_CUDA_Error(stdout,"hipSetDevice");
         hipHostMalloc((void**)&tmp[thread],max_mapped_memory_per_thread);  
         Check_CUDA_Error(outfile,"cpu tmp");
         hipMalloc((void**)&gpuarray[thread],gpumemory-extraroom);
         Check_CUDA_Error(outfile,"gpu memory");
     }
     fprintf(outfile,"done.\n");
     fprintf(outfile,"\n");
     fflush(outfile);

     // some cpu memory for cores to use when stealing gpu work 
     cpuarray = (double**)malloc(num_cpus*sizeof(double*));
     for (long int i=0; i<num_cpus; i++){
         // TODO: need to be more intelligent about this...
         cpuarray[i] = (double*)malloc(3*max_mapped_memory_per_thread+20*max_mapped_memory_per_thread/30);
     }
  }
}
/*===================================================================

  free gpu and mapped cpu memory

===================================================================*/
void GPUHelper::CudaFinalizeGPU(Options&options){
  if (num_gpus>0){
     #pragma omp parallel for schedule (static) num_threads(num_gpus)
     for (long int i=0; i<num_gpus; i++){
         long int thread = 0;
         #ifdef _OPENMP
           thread = omp_get_thread_num();
         #endif
         hipSetDevice(thread);
         Check_CUDA_Error(stdout,"hipSetDevice (free)");
         hipHostFree(tmp[thread]);
         Check_CUDA_Error(outfile,"cpu tmp (free)");
         hipFree(gpuarray[thread]);
         Check_CUDA_Error(outfile,"gpu memory (free)");
     }
     free(tmp);
     free(gpuarray);
     for (long int i=0; i<num_cpus; i++){
         free(cpuarray[i]);
     }
     free(cpuarray);
  }
}

/**
 * dgemm assuming no tiling is necessary
 */
void GPUHelper::GPU_DGEMM(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){
  double*gpuA,*gpuB,*gpuC;
  hipMalloc((void**)&gpuA,m*k*sizeof(double));
  hipMalloc((void**)&gpuB,n*k*sizeof(double));
  hipMalloc((void**)&gpuC,m*n*sizeof(double));
  hipMemcpy(gpuA,A,m*k*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(gpuB,B,n*k*sizeof(double),hipMemcpyHostToDevice);
  hipblasDgemm(transa,transb,m,n,k,alpha,gpuA,lda,gpuB,ldb,beta,gpuC,ldc);
  hipMemcpy(C,gpuC,m*n*sizeof(double),hipMemcpyDeviceToHost);
  hipFree(gpuA);
  hipFree(gpuB);
  hipFree(gpuC);
}
/**
 * dgemm using a 2-dimensional tile - threaded versions for multiple gpus
 */
void GPUHelper::GPU_DGEMM_2DTile_nn_threaded_WithCpuStealing(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);
  //Tiling((gpumemory-extraroom)/8L,max_mapped_memory/num_gpus/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel num_threads(num_gpus+num_cpus)
  {

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  double*gpuA,*gpuB,*gpuC;
  // pointers to gpu memory
  if (thread<num_gpus){
     gpuA = gpuarray[thread];
     gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
     gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;
  }
  // pointers to cpu memory
  else {
     gpuA = cpuarray[thread-num_gpus];
     gpuB = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK;
     gpuC = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK+tilesizeNprime*tilesizeK;
  }

  // cpu takes some of the 'N' tile
  if (StolenDimension=='N'){
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tk=0; tk<ntilesK; tk++){

             // this is for the gpus:
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
             
                for (long int tn=0; tn<ntilesN; tn++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesN[tn]; i++){
                        F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             // this if for any cpu cores that might be helping:
             else{
                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,gpuA+i*tilesizesM[tm],1);
                }

                for (long int tn=0; tn<ntilesNprime; tn++){
                    if ((tm*ntilesNprime+tn)%num_cpus + num_gpus!=thread) continue;
                    for (long int i=0; i<tilesizesNprime[tn]; i++){
                        F_DCOPY(tilesizesK[tk],B+(NprimeOffSet+i+tn*tilesizeNprime)*ldb+tk*tilesizeK,1,gpuB+i*tilesizesK[tk],1);
                    }
                    F_DGEMM(transa,transb,tilesizesM[tm],tilesizesNprime[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    for (long int j=0; j<tilesizesNprime[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,gpuC+j*tilesizesM[tm],1,C+(NprimeOffSet+j+tn*tilesizeNprime)*ldc+tm*tilesizeM,1);
                    }
                }
             }
         }
     }   
  }   
  // cpu takes some of the 'M' tile
  else if (StolenDimension=='M'){
     for (long int tn=0; tn<ntilesN; tn++){
         for (long int tk=0; tk<ntilesK; tk++){

             // this is for the gpus:
             if (thread<num_gpus){

                for (long int i=0; i<tilesizesN[tn]; i++){
                    F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                }
                hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
             
                for (long int tm=0; tm<ntilesM; tm++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             // this if for any cpu cores that might be helping:
             else{
                for (long int i=0; i<tilesizesN[tn]; i++){
                    F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,gpuB+i*tilesizesK[tk],1);
                }
             
                for (long int tm=0; tm<ntilesMprime; tm++){
                    if ((tm*ntilesN+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesMprime[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeMprime+MprimeOffSet,1,gpuA+i*tilesizesMprime[tm],1);
                    }
                    F_DGEMM(transa,transb,tilesizesMprime[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesMprime[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesMprime[tm]);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesMprime[tm],1.0,gpuC+j*tilesizesMprime[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeMprime+MprimeOffSet,1);
                    }
                }
             }
         }
     }   
  }
  else{
     if (thread<num_gpus){
        for (long int tm=0; tm<ntilesM; tm++){
            for (long int tk=0; tk<ntilesK; tk++){

                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                
                for (long int tn=0; tn<ntilesN; tn++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesN[tn]; i++){
                        F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
            }
        }
     }
  }

  }
  free(tilesizesMprime);
  free(tilesizesNprime);
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
/**
 * dgemm using a 2-dimensional tile - threaded versions for multiple gpus
 */
void GPUHelper::GPU_DGEMM_2DTile_nn_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      long int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      for (long int tk=0; tk<ntilesK; tk++){

          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesN[tn]; i++){
              F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
      }
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_nn(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc,int thread){

  TilingNoThread((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  for (long int mn=0; mn<ntilesM*ntilesN; mn++){

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      for (long int tk=0; tk<ntilesK; tk++){

          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesN[tn]; i++){
              F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
      }
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_nt_threaded_WithCpuStealing(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  //TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);
  TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;


  #pragma omp parallel num_threads(num_gpus+num_cpus)
  {

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  double*gpuA,*gpuB,*gpuC;

  // pointers to gpu memory
  if (thread<num_gpus){
     gpuA = gpuarray[thread];
     gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
     gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;
  }
  // pointers to cpu memory
  else {
     gpuA = cpuarray[thread-num_gpus];
     gpuB = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK;
     gpuC = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK+tilesizeNprime*tilesizeK;
  }

  // cpu takes some of the 'N' tile
  if (StolenDimension=='N'){
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){

                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                for (long int tn=0; tn<ntilesN; tn++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
                   hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                   for (long int j=0; j<tilesizesN[tn]; j++){
                       F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                   }
                }

             }
             else{

                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,gpuA+i*tilesizesM[tm],1);
                }

                for (long int tn=0; tn<ntilesNprime; tn++){
                    if ((tm*ntilesNprime+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesNprime[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeNprime+NprimeOffSet,1,gpuB+i*tilesizesNprime[tn],1);
                    }
                    F_DGEMM(transa,transb,tilesizesM[tm],tilesizesNprime[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesNprime[tn],0.0,gpuC,tilesizesM[tm]);
                    for (long int j=0; j<tilesizesNprime[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,gpuC+j*tilesizesM[tm],1,C+(j+tn*tilesizeNprime+NprimeOffSet)*ldc+tm*tilesizeM,1);
                    }
                }

             }
         }
     }
  }
  else if (StolenDimension=='M'){
     for (long int tn=0; tn<ntilesN; tn++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){

                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                }
                hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                for (long int tm=0; tm<ntilesM; tm++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }

             }
             else{

                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,gpuB+i*tilesizesN[tn],1);
                }

                for (long int tm=0; tm<ntilesMprime; tm++){
                    if ((tm*ntilesN+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesMprime[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeMprime+MprimeOffSet,1,gpuA+i*tilesizesMprime[tm],1);
                    }

                    F_DGEMM(transa,transb,tilesizesMprime[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesMprime[tm],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesMprime[tm]);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesMprime[tm],1.0,gpuC+j*tilesizesMprime[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeMprime+MprimeOffSet,1);
                    }
                }

             }
         }
     }
  }
  else{
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tn=0; tn<ntilesN; tn++){
             if (thread<num_gpus){
                if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
                for (long int tk=0; tk<ntilesK; tk++){
                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
                }
                hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                for (long int j=0; j<tilesizesN[tn]; j++){
                    F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                }

             }
         }
     }
  }
  }

  free(tilesizesNprime);
  free(tilesizesMprime);
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_nt_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      long int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
      }
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_nt(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc,int thread){

  TilingNoThread((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  for (long int mn=0; mn<ntilesM*ntilesN; mn++){

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
      }
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tn_threaded_WithCpuStealing(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel num_threads(num_gpus+num_cpus)
  {

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  double*gpuA,*gpuB,*gpuC;

  // pointers to gpu memory
  if (thread<num_gpus){
     gpuA = gpuarray[thread];
     gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
     gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;
  }
  // pointers to cpu memory
  else {
     gpuA = cpuarray[thread-num_gpus];
     gpuB = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK;
     gpuC = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK+tilesizeNprime*tilesizeK;
  }

  // cpu takes some of the 'N' tile
  StolenDimension=' ';
  if (StolenDimension=='N'){
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesM[tm]; i++){
                    F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                for (long int tn=0; tn<ntilesN; tn++){

                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesN[tn]; i++){
                        F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             else{
                for (long int i=0; i<tilesizesM[tm]; i++){
                    F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,gpuA+i*tilesizesK[tk],1);
                }

                for (long int tn=0; tn<ntilesNprime; tn++){

                    if ((tm*ntilesNprime+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesNprime[tn]; i++){
                        F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeNprime+NprimeOffSet)*ldb+tk*tilesizeK,1,gpuB+i*tilesizesK[tk],1);
                    }
                    F_DGEMM(transa,transb,tilesizesM[tm],tilesizesNprime[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    for (long int j=0; j<tilesizesNprime[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,gpuC+j*tilesizesM[tm],1,C+(j+tn*tilesizeNprime+NprimeOffSet)*ldc+tm*tilesizeM,1);
                    }
                }
             }
         }
     }
  }
  else if (StolenDimension=='M'){
     for (long int tn=0; tn<ntilesN; tn++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesN[tn]; i++){
                    F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                }
                hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                for (long int tm=0; tm<ntilesM; tm++){

                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesM[tm]; i++){
                        F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             else{
                for (long int i=0; i<tilesizesN[tn]; i++){
                    F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,gpuB+i*tilesizesK[tk],1);
                }

                for (long int tm=0; tm<ntilesMprime; tm++){

                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesMprime[tm]; i++){
                        F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeMprime+MprimeOffSet)*lda+tk*tilesizeK,1,gpuA+i*tilesizesK[tk],1);
                    }
                    F_DGEMM(transa,transb,tilesizesMprime[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesMprime[tm]);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesMprime[tm],1.0,gpuC+j*tilesizesMprime[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeMprime+MprimeOffSet,1);
                    }
                }
             }
         }
     }
  }
  else{
     if (thread<num_gpus){
        for (long int tm=0; tm<ntilesM; tm++){
            for (long int tn=0; tn<ntilesN; tn++){
                hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
                for (long int tk=0; tk<ntilesK; tk++){
                    for (long int i=0; i<tilesizesM[tm]; i++){
                        F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    for (long int i=0; i<tilesizesN[tn]; i++){
                        F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
                }
                hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                for (long int j=0; j<tilesizesN[tn]; j++){
                    F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                }

            }
        }
     }
  }


  }
  free(tilesizesMprime);
  free(tilesizesNprime);
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tn_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      long int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesM[tm]; i++){
              F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesN[tn]; i++){
              F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
      }
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tn(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc,int thread){

  TilingNoThread((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  for (long int mn=0; mn<ntilesM*ntilesN; mn++){

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesM[tm]; i++){
              F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesN[tn]; i++){
              F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
      }
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tt_threaded_WithCpuStealing(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;


  #pragma omp parallel num_threads(num_gpus+num_cpus)
  {

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  double*gpuA,*gpuB,*gpuC;

  // pointers to gpu memory
  if (thread<num_gpus){
     gpuA = gpuarray[thread];
     gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
     gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;
  }
  // pointers to cpu memory
  else {
     gpuA = cpuarray[thread-num_gpus];
     gpuB = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK;
     gpuC = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK+tilesizeNprime*tilesizeK;
  }

  // cpu takes some of the 'N' tile
  StolenDimension=' ';
  if (StolenDimension=='N'){
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesM[tm]; i++){
                    F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                for (long int tn=0; tn<ntilesN; tn++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             else{
                for (long int i=0; i<tilesizesM[tm]; i++){
                    F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,gpuA+i*tilesizesK[tk],1);
                }
                for (long int tn=0; tn<ntilesNprime; tn++){
                    if ((tm*ntilesNprime+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesNprime[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeNprime+NprimeOffSet,1,gpuB+i*tilesizesN[tn],1);
                    }
                    F_DGEMM(transa,transb,tilesizesM[tm],tilesizesNprime[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesNprime[tn],0.0,gpuC,tilesizesM[tm]);
                    for (long int j=0; j<tilesizesNprime[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,gpuC+j*tilesizesM[tm],1,C+(j+tn*tilesizeNprime+NprimeOffSet)*ldc+tm*tilesizeM,1);
                    }
                }
             }
         }
     }
  }
  else if (StolenDimension=='M'){
     for (long int tn=0; tn<ntilesN; tn++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                }
                hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                for (long int tm=0; tm<ntilesM; tm++){

                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesM[tm]; i++){
                        F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             else{
                for (long int i=0; i<tilesizesK[tk]; i++){
                    F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,gpuB+i*tilesizesN[tn],1);
                }
                for (long int tm=0; tm<ntilesMprime; tm++){

                    if ((tm*ntilesN+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesMprime[tm]; i++){
                        F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeMprime+MprimeOffSet)*lda+tk*tilesizeK,1,gpuA+i*tilesizesK[tk],1);
                    }
                    F_DGEMM(transa,transb,tilesizesMprime[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesMprime[tm]);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        F_DAXPY(tilesizesMprime[tm],1.0,gpuC+j*tilesizesMprime[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeMprime+MprimeOffSet,1);
                    }
                }
             }
         }
     }
  }
  else{
     if (thread<num_gpus){
        for (long int tm=0; tm<ntilesM; tm++){
            for (long int tn=0; tn<ntilesN; tn++){
                hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
                for (long int tk=0; tk<ntilesK; tk++){
                    for (long int i=0; i<tilesizesM[tm]; i++){
                        F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    for (long int i=0; i<tilesizesK[tk]; i++){
                        F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
                }
                hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                for (long int j=0; j<tilesizesN[tn]; j++){
                    F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                }
            }
        }
     }
  }

  }

  free(tilesizesMprime);
  free(tilesizesNprime);
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tt_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      long int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesM[tm]; i++){
              F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
      }
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tt(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc,int thread){

  TilingNoThread((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  for (long int mn=0; mn<ntilesM*ntilesN; mn++){

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesM[tm]; i++){
              F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
      }
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}

void GPUHelper::TilingNoThread(long int mem1,long int mem2,long int m,long int n,long int k){

  // first tile according to how much space is on gpu
  tilesizeN = n;
  tilesizeM = m;
  tilesizeK = k;
  ntilesM=ntilesN=ntilesK=1L;
  while(tilesizeN*tilesizeM+tilesizeK*(tilesizeN+tilesizeM)>mem1){
     if (tilesizeN>tilesizeM){
        if (tilesizeN>tilesizeK){
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
     }
     else{
        if (tilesizeM>tilesizeK){
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
        else{
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
     }
  }

  // ensure each block of A, B, and C will fit in the temporary CPU buffer
  while(tilesizeN*tilesizeM>mem2){
     if (tilesizeN>tilesizeM){
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
  }

  while(tilesizeN*tilesizeK>mem2){
     if (tilesizeN>tilesizeK){
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        ntilesK++;
        tilesizeK = k/ntilesK;
        if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
     }
  }
  while(tilesizeK*tilesizeM>mem2){
     if (tilesizeK>tilesizeM){
        ntilesK++;
        tilesizeK = k/ntilesK;
        if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
     }
     else{
        ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
  }

  lasttileN = n - (ntilesN-1L)*tilesizeN;
  lasttileM = m - (ntilesM-1L)*tilesizeM;
  lasttileK = k - (ntilesK-1L)*tilesizeK;

  tilesizesM = (long int*)malloc(ntilesM*sizeof(long int));
  tilesizesN = (long int*)malloc(ntilesN*sizeof(long int));
  tilesizesK = (long int*)malloc(ntilesK*sizeof(long int));
  for (long int i=0; i<ntilesM-1L; i++) tilesizesM[i] = tilesizeM;
  for (long int i=0; i<ntilesN-1L; i++) tilesizesN[i] = tilesizeN;
  for (long int i=0; i<ntilesK-1L; i++) tilesizesK[i] = tilesizeK;
  tilesizesM[ntilesM-1L] = lasttileM;
  tilesizesN[ntilesN-1L] = lasttileN;
  tilesizesK[ntilesK-1L] = lasttileK;

  //printf("%5li %5li %5li (n^5)\n",ntilesM,ntilesN,ntilesK);fflush(stdout);

}
void GPUHelper::Tiling(long int mem1,long int mem2,long int m,long int n,long int k){

  // first tile according to how much space is on gpu
  tilesizeN = n;
  tilesizeM = m;
  tilesizeK = k;
  ntilesM=ntilesN=ntilesK=1L;
  while(tilesizeN*tilesizeM+tilesizeK*(tilesizeN+tilesizeM)>mem1){
     if (ntilesN*ntilesM<num_gpus){
        if (tilesizeN>tilesizeM){
           ntilesN++;
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesM++;
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
     }
     else{
        if (tilesizeN>tilesizeM){
           if (tilesizeN>tilesizeK){
              ntilesN++;
              tilesizeN = n/ntilesN;
              if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
           }
           else{
              ntilesK++;
              tilesizeK = k/ntilesK;
              if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
           }
        }
        else{
           if (tilesizeM>tilesizeK){
              ntilesM++;
              tilesizeM = m/ntilesM;
              if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
           }
           else{
              ntilesK++;
              tilesizeK = k/ntilesK;
              if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
           }
        }
     }
  }

  // ensure each block of A, B, and C will fit in the temporary CPU buffer
  while(tilesizeN*tilesizeM>mem2){
     if (ntilesN*ntilesM<num_gpus){
        if (tilesizeN>tilesizeM){
           ntilesN++;
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesM++;
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
     }
     else{
        if (tilesizeN>tilesizeM){
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
     }
  }

  while(tilesizeN*tilesizeK>mem2){
     if (ntilesN*ntilesM<num_gpus){
        ntilesN++;
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        if (tilesizeN>tilesizeK){
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
     }
  }
  while(tilesizeK*tilesizeM>mem2){
     if (ntilesN*ntilesM<num_gpus){
        ntilesM++;
        ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
     else{
        if (tilesizeK>tilesizeM){
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
        else{
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
     }
  }

  // finally make sure that we've tiled enough so each gpu has something to do
  // TODO:  i need a way to make sure these end up 3:1, not 2:2
  //        ...should probably be more general than this...
  while(ntilesN*ntilesM<num_gpus){
     if (tilesizeN>tilesizeM){
        ntilesN++;
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        ntilesM++;
        ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
  }

  lasttileN = n - (ntilesN-1L)*tilesizeN;
  lasttileM = m - (ntilesM-1L)*tilesizeM;
  lasttileK = k - (ntilesK-1L)*tilesizeK;

  tilesizesM = (long int*)malloc(ntilesM*sizeof(long int));
  tilesizesN = (long int*)malloc(ntilesN*sizeof(long int));
  tilesizesK = (long int*)malloc(ntilesK*sizeof(long int));
  for (long int i=0; i<ntilesM-1L; i++) tilesizesM[i] = tilesizeM;
  for (long int i=0; i<ntilesN-1L; i++) tilesizesN[i] = tilesizeN;
  for (long int i=0; i<ntilesK-1L; i++) tilesizesK[i] = tilesizeK;
  tilesizesM[ntilesM-1L] = lasttileM;
  tilesizesN[ntilesN-1L] = lasttileN;
  tilesizesK[ntilesK-1L] = lasttileK;

  //printf("%5li %5li %5li\n",ntilesM,ntilesN,ntilesK);fflush(stdout);

}
void GPUHelper::TilingWithCpuStealing(long int mem1,long int mem2,long int m,long int n,long int k){
  // compute normal tiling
  Tiling(mem1,mem2,m,n,k);
  // take a slice of the larger of m or n for the cpu
  
  // first let's just try taking a sliver of the last tile of N...
  ntilesNprime = num_cpus;
  ntilesMprime = num_cpus;
  tilesizesNprime = (long int*)malloc(ntilesNprime*sizeof(long int));
  tilesizesMprime = (long int*)malloc(ntilesMprime*sizeof(long int));

  // which dimension will cpu work on?
  if (tilesizeN>tilesizeM){
     // assume the gpu is ~30x faster than a single core:
     if (tilesizeN<30){
        StolenDimension = ' ';
        return;
     }
     tilesizeNprime = tilesizeN/30;

     StolenDimension = 'N';

     // need to figure out new tiles in N (and might as well make them even)
     // TODO: should make these multiples of the warp size, too
     long int newn = n-num_cpus*tilesizeNprime;
     tilesizeN = newn/ntilesN-1;
     lasttileN = tilesizeN;
     for (long int i=0; i<ntilesN; i++)
         tilesizesN[i] = tilesizeN;

     // redo Nprime's numbers
     ntilesNprime   = num_cpus;
     NprimeOffSet   = ntilesN*tilesizeN;
     tilesizeNprime = (n - NprimeOffSet)/ntilesNprime;
     if (tilesizeNprime*ntilesNprime<(n-NprimeOffSet)) tilesizeNprime++;
     lasttileNprime = (n - NprimeOffSet)-(ntilesNprime-1)*tilesizeNprime;

     for (long int i=0; i<ntilesNprime-1; i++) tilesizesNprime[i] = tilesizeNprime;
     tilesizesNprime[ntilesNprime-1] = lasttileNprime;

     // set this just for memory mapping
     lasttileMprime = 0;
     tilesizesMprime[0] = lasttileMprime;
     tilesizeMprime = tilesizeM;
  }
  // do M instead:
  else{
     // assume the gpu is ~30x faster than a single core:
     if (tilesizeM<30){
        StolenDimension = ' ';
        return;
     }
     tilesizeMprime = tilesizeM/30;

     StolenDimension = 'M';

     // need to figure out new tiles in N (and might as well make them even)
     // TODO: should make these multiples of the warp size, too
     long int newm = m-num_cpus*tilesizeMprime;
     tilesizeM = newm/ntilesM-1;
     lasttileM = tilesizeM;
     for (long int i=0; i<ntilesM; i++)
         tilesizesM[i] = tilesizeM;

     // redo Mprime's numbers
     ntilesMprime   = num_cpus;
     MprimeOffSet   = ntilesM*tilesizeM;
     tilesizeMprime = (m - MprimeOffSet)/ntilesMprime;
     if (tilesizeMprime*ntilesMprime<(m-MprimeOffSet)) tilesizeMprime++;
     lasttileMprime = (m - MprimeOffSet)-(ntilesMprime-1)*tilesizeMprime;

     for (long int i=0; i<ntilesMprime-1; i++) tilesizesMprime[i] = tilesizeMprime;
     tilesizesMprime[ntilesMprime-1] = lasttileMprime;

     // set this just for memory mapping
     lasttileNprime = 0;
     tilesizesNprime[0] = lasttileNprime;
     tilesizeNprime = tilesizeN;

     //printf("hey the tile is %5li (%5li) %5li %5li\n",tilesizeMprime,m,ntilesM,tilesizeM);fflush(stdout);
  }
}

}//end of namespace psi


