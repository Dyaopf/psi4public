#include <libplugin/plugin.h>
#include"psi4-dec.h"
#include<boost/shared_ptr.hpp>
#include<liboptions/liboptions.h>
#include<libtrans/integraltransform.h>
#include<libtrans/mospace.h>
#include<libmints/matrix.h>
#include<libmints/vector.h>
#include<libchkpt/chkpt.h>
#include<libiwl/iwl.h>
#include <libpsio/psio.hpp>

#include"globals.h"
#include"blas.h"
#include"gpuhelper.h"
#include"gpuonly.h"
#ifdef _OPENMP
  #include<omp.h>
#endif


using namespace psi;
using namespace boost;

namespace psi{

GPUHelper::GPUHelper()
{}
GPUHelper::~GPUHelper()
{}

inline void GPUHelper::Check_CUDA_Error(FILE*fp,const char *message){
  hipError_t error = hipGetLastError();
  if (error!=hipSuccess) {
     fprintf(fp,"\n  ERROR: %s: %s\n\n", message, hipGetErrorString(error) );
     fflush(fp);
     exit(-1);
  }
}

/*===================================================================

  initialize cublas and get device properties

===================================================================*/
void GPUHelper::CudaInit(Options&options){

  num_gpus=gpumemory=extraroom=0;
  hipGetDeviceCount(&num_gpus);
  if (num_gpus>0){
     cublasInit();
     struct hipDeviceProp_t cudaProp;
     int gpu_id;
     hipGetDevice(&gpu_id);
     hipGetDeviceProperties( &cudaProp,gpu_id );
     fprintf(outfile,
       "\n  _________________________________________________________\n");
     fprintf(outfile,"  CUDA device properties:\n");
     fprintf(outfile,"  name:                 %20s\n",cudaProp.name);
     fprintf(outfile,"  major version:        %20d\n",cudaProp.major);
     fprintf(outfile,"  minor version:        %20d\n",cudaProp.minor);
     fprintf(outfile,"  canMapHostMemory:     %20d\n",cudaProp.canMapHostMemory);
     fprintf(outfile,"  totalGlobalMem:       %20lu mb\n",
       cudaProp.totalGlobalMem/(1024*1024));
     fprintf(outfile,"  sharedMemPerBlock:    %20lu\n",cudaProp.sharedMemPerBlock);
     fprintf(outfile,"  clockRate:            %20.3f ghz\n",
       cudaProp.clockRate/1.0e6);
     fprintf(outfile,"  regsPerBlock:         %20d\n",cudaProp.regsPerBlock);
     fprintf(outfile,"  warpSize:             %20d\n",cudaProp.warpSize);
     fprintf(outfile,"  maxThreadsPerBlock:   %20d\n",cudaProp.maxThreadsPerBlock);
     fprintf(outfile,
       "  _________________________________________________________\n\n");
     fflush(outfile);

     gpumemory = cudaProp.totalGlobalMem;

     extraroom = 350*1024*1024;
     
     hipDeviceReset();

     // default memory for mapped cpu memory is the sum of all gpu memory
     max_mapped_memory = num_gpus * (gpumemory-extraroom);
     if (options["MAX_MAPPED_MEMORY"].has_changed()){
        ULI temp_mem = options.get_int("MAX_MAPPED_MEMORY");
        temp_mem *= 1024*1024;
        if (temp_mem<max_mapped_memory)
           max_mapped_memory = options.get_int("MAX_MAPPED_MEMORY");
     }

     fprintf(outfile,"\n");
     fprintf(outfile,"  allocating gpu memory...");
     fflush(outfile);
     tmp = (double**)malloc(num_gpus*sizeof(double*));
     gpuarray = (double**)malloc(num_gpus*sizeof(double*));
     #pragma omp parallel for schedule (static) num_threads(num_gpus)
     for (int i=0; i<num_gpus; i++){
         int thread = 0;
         #ifdef _OPENMP
           thread = omp_get_thread_num();
         #endif
         hipSetDevice(thread);
         Check_CUDA_Error(stdout,"hipSetDevice");
         hipHostMalloc((void**)&tmp[thread],max_mapped_memory/num_gpus);  
         Check_CUDA_Error(outfile,"cpu tmp");
         hipMalloc((void**)&gpuarray[thread],gpumemory-extraroom);
         Check_CUDA_Error(outfile,"gpu memory");
     }
     fprintf(outfile,"done.\n");
     fprintf(outfile,"\n");
     fflush(outfile);
  }
}

/**
 * dgemm assuming no tiling is necessary
 */
void GPUHelper::GPU_DGEMM(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){
  double*gpuA,*gpuB,*gpuC;
  hipMalloc((void**)&gpuA,m*k*sizeof(double));
  hipMalloc((void**)&gpuB,n*k*sizeof(double));
  hipMalloc((void**)&gpuC,m*n*sizeof(double));
  hipMemcpy(gpuA,A,m*k*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(gpuB,B,n*k*sizeof(double),hipMemcpyHostToDevice);
  hipblasDgemm(transa,transb,m,n,k,alpha,gpuA,lda,gpuB,ldb,beta,gpuC,ldc);
  hipMemcpy(C,gpuC,m*n*sizeof(double),hipMemcpyDeviceToHost);
  hipFree(gpuA);
  hipFree(gpuB);
  hipFree(gpuC);
}
/**
 * dgemm using a 2-dimensional tile.
 */
void GPUHelper::GPUTiledDGEMM(char transa,char transb,long int m, long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){
  if (num_gpus<1){
     F_DGEMM(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
     return;
  }
  /* if (thread>=num_gpus){
     F_DGEMM(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
  }*/
  if (transa=='n'){
     if (transb=='n'){
        GPU_DGEMM_2DTile_nn_threaded(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
        //F_DGEMM(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
     }
     else{
        GPU_DGEMM_2DTile_nt_threaded(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
        //F_DGEMM(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
     }
  }
  else{
     if (transb=='n'){
        GPU_DGEMM_2DTile_tn_threaded(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
        //F_DGEMM(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
     }
     else{
        GPU_DGEMM_2DTile_tt_threaded(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
        //F_DGEMM(transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
     }
  }
}
/**
 * dgemm using a 2-dimensional tile - threaded versions for multiple gpus
 */
void GPUHelper::GPU_DGEMM_2DTile_nn_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8.,max_mapped_memory/num_gpus/8.,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      for (long int tk=0; tk<ntilesK; tk++){

          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesN[tn]; i++){
              F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
          hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
          for (long int j=0; j<tilesizesN[tn]; j++){
              F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
          }
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_nt_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8.,max_mapped_memory/num_gpus/8.,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
          hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
          for (long int j=0; j<tilesizesN[tn]; j++){
              F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
          }
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tn_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8.,max_mapped_memory/num_gpus/8.,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesM[tm]; i++){
              F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesN[tn]; i++){
              F_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
          hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
          for (long int j=0; j<tilesizesN[tn]; j++){
              F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
          }
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tt_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8.,max_mapped_memory/num_gpus/8.,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif

      // pointers to gpu memory
      double*gpuA = gpuarray[thread];
      double*gpuB = gpuarray[thread]+tilesizeM*tilesizeK;
      double*gpuC = gpuarray[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesM[tm]; i++){
              F_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesK[tk]; i++){
              F_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
          hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
          for (long int j=0; j<tilesizesN[tn]; j++){
              F_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
          }
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}

void GPUHelper::Tiling(double mem1,double mem2,long int m,long int n,long int k){

  // first tile according to how much space is on gpu
  tilesizeN = n;
  tilesizeM = m;
  tilesizeK = k;
  ntilesM=ntilesN=ntilesK=1;
  while(tilesizeN*tilesizeM+tilesizeK*(tilesizeN+tilesizeM)>mem1){
     if (tilesizeN>tilesizeM){
        if (tilesizeN>tilesizeK){
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
     }
     else{
        if (tilesizeM>tilesizeK){
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
        else{
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
     }
  }

  // ensure each block of A, B, and C will fit in the temporary CPU buffer
  while(tilesizeN*tilesizeM>mem2){
     if (tilesizeN>tilesizeM){
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
  }
  while(tilesizeN*tilesizeK>mem2){
     if (tilesizeN>tilesizeK){
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        ntilesK++;
        tilesizeK = k/ntilesK;
        if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
     }
  }
  while(tilesizeK*tilesizeM>mem2){
     if (tilesizeK>tilesizeM){
        ntilesK++;
        tilesizeK = k/ntilesK;
        if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
     }
     else{
        ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
  }

  // finally make sure that we've tiled enough so each gpu has something to do
  // TODO:  i need a way to make sure these end up 3:1, not 2:2
  //        ...should probably be more general than this...
  while(ntilesN*ntilesM<num_gpus){
     if (tilesizeN>tilesizeM){
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
  }

  lasttileN = n - (ntilesN-1)*tilesizeN;
  lasttileM = m - (ntilesM-1)*tilesizeM;
  lasttileK = k - (ntilesK-1)*tilesizeK;

  tilesizesM = (long int*)malloc(ntilesM*sizeof(long int));
  tilesizesN = (long int*)malloc(ntilesN*sizeof(long int));
  tilesizesK = (long int*)malloc(ntilesK*sizeof(long int));
  for (long int i=0; i<ntilesM-1; i++) tilesizesM[i] = tilesizeM;
  for (long int i=0; i<ntilesN-1; i++) tilesizesN[i] = tilesizeN;
  for (long int i=0; i<ntilesK-1; i++) tilesizesK[i] = tilesizeK;
  tilesizesM[ntilesM-1] = lasttileM;
  tilesizesN[ntilesN-1] = lasttileN;
  tilesizesK[ntilesK-1] = lasttileK;

}

}//end of namespace psi


